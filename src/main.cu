#include <stdio.h>
#include "../include/constants.h"
#include "../include/stencil_initializer.h"
#include "../include/stencil_error_checker.h"
#include "../include/stencil_utils.h"
#include "stencil_cpu.cu"
#include "stencil_dispatch.cu"
#include "omp.h"

int main(int argc, const char **argv) {
    float  *h_ref, *d_ref,
           *d_u1, *d_u2,
           milli;

    if (DEBUG) {
        h_ref = (float *)malloc(BYTES);
        char f[] = SOLUTION;
        if (!file_exists(f)) stencil_cpu();
        readSolution(h_ref);
    }

    d_ref = (float *)malloc(BYTES);

    initialize_host_region(d_ref);

    CU(hipMalloc((void **)&d_u1, BYTES));
    CU(hipMalloc((void **)&d_u2, BYTES));

    CU(hipMemcpy(d_u1, d_ref, BYTES, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CU(hipEventCreate(&start));
    CU(hipEventCreate(&stop));
    CU(hipEventRecord(start));

    dispatch_kernels(d_u1, d_u2);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CU(hipMemcpy(d_ref, d_u1, BYTES, hipMemcpyDeviceToHost));

    if (DEBUG) {
        check_domain_errors(h_ref, d_ref);
        free(h_ref);
    }

    print_program_info();
    printf("%.4f\n", milli); // Print execution time in ms

    free(d_ref);

    CU(hipFree(d_u1));
    CU(hipFree(d_u2));
    hipDeviceReset();
}
