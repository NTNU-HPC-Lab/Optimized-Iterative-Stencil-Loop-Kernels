#include "hip/hip_runtime.h"
#include "../include/constants.h"
#include "hip/hip_cooperative_groups.h"
#include "stencils.cu"
#include "prefetch_smem.cu"
#include "stencils_border_check.cu"
using namespace cooperative_groups;

__global__ void smem_padded_2d(float* __restrict__ d_u1,
                               float* __restrict__ d_u2,
                               unsigned int jstart,
                               unsigned int jend)
{
    extern __shared__ float smem[];
    unsigned int i, j, idx, sidx;
    i  = threadIdx.x + blockIdx.x*BLOCK_X*UNROLL_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;
    idx = i + j*NX;
    sidx = (threadIdx.x + STENCIL_DEPTH) + (threadIdx.y + STENCIL_DEPTH)*SMEM_P_X;
    if (check_domain_border_2d(i, j, jstart, jend))
        prefetch(smem, d_u1, 0, i, j, idx, sidx, jstart, jend);
    this_thread_block().sync();
    if (check_stencil_border_2d(i, j, jstart, jend))
        smem_padded_stencil(smem, d_u2, idx, sidx);
}

__global__ void smem_padded_unroll_2d(float* __restrict__ d_u1,
                                      float* __restrict__ d_u2,
                                      unsigned int jstart,
                                      unsigned int jend)
{
    extern __shared__ float smem[];
    unsigned int i, j, si, sj, i_off, si_off, u, idx, sidx;
    i  = threadIdx.x + blockIdx.x*BLOCK_X*UNROLL_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;
    si = threadIdx.x + STENCIL_DEPTH;
    sj = threadIdx.y + STENCIL_DEPTH;
#pragma unroll
    for (u=0; u<UNROLL_X; u++) {
        i_off  = i+u*BLOCK_X;
        si_off = si+u*BLOCK_X;
        idx    = i_off+j*NX;
        sidx   = si_off+sj*SMEM_P_X;
        if (check_domain_border_2d(i_off, j, jstart, jend))
            prefetch(smem, d_u1, u, i_off, j, idx, sidx, jstart, jend);
    }
    this_thread_block().sync();
#pragma unroll
    for (u=0; u<UNROLL_X; u++) {
        i_off  = i+u*BLOCK_X;
        idx    = i_off+j*NX;
        si_off = si+u*BLOCK_X;
        sidx   = si_off+sj*SMEM_P_X;
        if (check_stencil_border_2d(i_off, j, jstart, jend))
            smem_padded_stencil(smem, d_u2, idx, sidx);
    }
}
