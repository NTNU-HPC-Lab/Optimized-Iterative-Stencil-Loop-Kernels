#include "hip/hip_runtime.h"
#include "../include/constants.h"
#include "hip/hip_cooperative_groups.h"
#include "stencils.cu"
#include "prefetch_smem.cu"
#include "stencil_border_check.cu"
using namespace cooperative_groups;

__global__ void smem_register_3d(float* __restrict__ d_u1,
                                 float* __restrict__ d_u2,
                                 unsigned int kstart,
                                 unsigned int kend)
{
    unsigned int i, j, k, idx, sidx;
    extern __shared__ float smem[];
    float yval[REG_SIZE];
    i  = threadIdx.x + blockIdx.x*blockDim.x;
    j  = threadIdx.y + blockIdx.y*blockDim.y;
    k  = threadIdx.z + blockIdx.z*blockDim.z;
    idx = i + j*NX + k*NX*NY;
    sidx = (threadIdx.x + RADIUS)
         + (threadIdx.y + RADIUS)*SMEM_P_X
         + (threadIdx.z)*SMEM_P_X*SMEM_P_Y;
    if (check_domain_border_3d(i, j, k, kstart, kend))
        prefetch_register_3d(smem, d_u1, yval, 0, idx, sidx, i, j, k, kstart, kend);
    this_thread_block().sync();
    if (check_stencil_border_3d(i, j, k, kstart, kend))
        smem_register_stencil(smem, d_u2, yval, sidx, idx);
}

__global__ void smem_register_unroll_3d(float* __restrict__ d_u1,
                                        float* __restrict__ d_u2,
                                        unsigned int kstart,
                                        unsigned int kend)
{
    unsigned int i, j, k, s, si, sj, i_off, si_off, idx, sidx;
    extern __shared__ float smem[];
    float yval[COARSEN_X][REG_SIZE];
    i  = threadIdx.x + blockIdx.x*blockDim.x*COARSEN_X;
    j  = threadIdx.y + blockIdx.y*blockDim.y;
    k  = threadIdx.z + blockIdx.z*blockDim.z;
    si = threadIdx.x + RADIUS;
    sj = threadIdx.y + RADIUS;
#pragma unroll
    for (s=0; s<COARSEN_X; s++) {
        i_off = i + s*blockDim.x;
        idx = i_off + j*NX + k*NX*NY;
        si_off = si + s*blockDim.x;
        sidx = si_off + sj*SMEM_P_X + threadIdx.z*SMEM_P_X*SMEM_P_Y;
        if (check_domain_border_3d(i_off, j, k, kstart, kend))
            prefetch_register_3d(smem, d_u1, yval[s], s, idx, sidx, i_off, j, k, kstart, kend);
    }
    this_thread_block().sync();
#pragma unroll
    for (s=0; s<COARSEN_X; s++) {
        i_off = i + s*blockDim.x;
        idx = i_off + j*NX + k*NX*NY;
        si_off = si + s*blockDim.x;
        sidx = si_off + sj*SMEM_P_X + threadIdx.z*SMEM_P_X*SMEM_P_Y;
        if (check_stencil_border_3d(i_off, j, k, kstart, kend))
            smem_register_stencil(smem, d_u2, yval[s], sidx, idx);
    }
}

__global__ void smem_register_2d(float* __restrict__ d_u1,
                                 float* __restrict__ d_u2,
                                 unsigned int jstart,
                                 unsigned int jend)
{
    unsigned int i, j, idx, sidx;
    extern __shared__ float smem[];
    float yval[REG_SIZE];
    i  = threadIdx.x + blockIdx.x*blockDim.x;
    j  = threadIdx.y + blockIdx.y*blockDim.y;
    idx = i + j*NX;
    sidx = (threadIdx.x + RADIUS) + threadIdx.y*SMEM_P_X;
    if (check_domain_border_2d(i, j, jstart, jend))
        prefetch_register(smem, d_u1, yval, 0, idx, sidx, i, j, jstart, jend);
    this_thread_block().sync();
    if (check_stencil_border_2d(i, j, jstart, jend))
        smem_register_stencil(smem, d_u2, yval, sidx, idx);
}

__global__ void smem_register_unroll_2d(float* __restrict__ d_u1,
                                        float* __restrict__ d_u2,
                                        unsigned int jstart,
                                        unsigned int jend)
{
    unsigned int i, j, s, si, i_off, idx, sidx, ioff;
    extern __shared__ float smem[];
    float yval[COARSEN_X][REG_SIZE], *ys;
    i  = threadIdx.x + blockIdx.x*blockDim.x*COARSEN_X;
    j  = threadIdx.y + blockIdx.y*blockDim.y;
    idx = i + j*NX;
    si = threadIdx.x + RADIUS;
#pragma unroll
    for (s=0; s<COARSEN_X; s++) {
        ioff = s*blockDim.x;
        i_off = i + ioff;
        idx = i_off + j*NX;
        sidx = (si+ioff) + threadIdx.y*SMEM_P_X;
        ys = yval[s];
        if (check_domain_border_2d(i_off, j, jstart, jend))
            prefetch_register(smem, d_u1, ys, s, idx, sidx, i_off, j, jstart, jend);
    }
    this_thread_block().sync();
#pragma unroll
    for (s=0; s<COARSEN_X; s++) {
        ioff = s*blockDim.x;
        i_off = i + ioff;
        idx = i_off + j*NX;
        sidx = (si+ioff) + threadIdx.y*SMEM_P_X;
        ys = yval[s];
        if (check_stencil_border_2d(i_off, j, jstart, jend))
            smem_register_stencil(smem, d_u2, ys, sidx, idx);
    }
}

