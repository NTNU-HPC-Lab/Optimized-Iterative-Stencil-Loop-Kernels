#include "hip/hip_runtime.h"
#include "../include/constants.h"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

__device__ float smem_stencil(float* smem, float* d_u1, unsigned int sidx, unsigned int idx, float u) {
    unsigned int d;
#pragma unroll
    for (d=STENCIL_DEPTH; d>=1; d--) {
        if (threadIdx.x >= d)        u += smem[sidx-d];
        else                         u += d_u1[idx-d];
    }
#pragma unroll
    for (d=1; d<=STENCIL_DEPTH; d++) {
        if (threadIdx.x+d < BLOCK_X) u += smem[sidx+d];
        else                         u += d_u1[idx+d];
    }
#if DIMENSIONS>1
#pragma unroll
    for (d=STENCIL_DEPTH; d>=1; d--) {
        if (threadIdx.y >= d)        u += smem[sidx-d*(BLOCK_X+SMEM_PAD)];
        else                         u += d_u1[idx-d*NX];
    }
#pragma unroll
    for (d=1; d<=STENCIL_DEPTH; d++) {
        if (threadIdx.y+d < BLOCK_Y) u += smem[sidx+d*(BLOCK_X+SMEM_PAD)];
        else                         u += d_u1[idx+d*NX];
    }
#endif
#if DIMENSIONS>2
#pragma unroll
    for (d=STENCIL_DEPTH; d>=1; d--) {
        if (threadIdx.z >= d)        u += smem[sidx-d*(BLOCK_X+SMEM_PAD)*BLOCK_Y];
        else                         u += d_u1[idx-d*NX*NY];
    }
#pragma unroll
    for (d=1; d<=STENCIL_DEPTH; d++) {
        if (threadIdx.z+d < BLOCK_Z) u += smem[sidx+d*(BLOCK_X+SMEM_PAD)*BLOCK_Y];
        else                         u += d_u1[idx+d*NX*NY];
    }
#endif
    return u;
}

__global__ void gpu_stencil_smem_3d(float* __restrict__ d_u1,
			            float* __restrict__ d_u2,
                                    unsigned int kstart,
                                    unsigned int kend)
{
    float u = 0.0f, u0;
    unsigned int   i, j, k, idx, sidx;
    extern __shared__ float smem[];
    i  = threadIdx.x + blockIdx.x*BLOCK_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;
    k  = threadIdx.z + blockIdx.z*BLOCK_Z;
    idx = i + j*NX + k*NX*NY;
    sidx = threadIdx.x + threadIdx.y*(BLOCK_X+SMEM_PAD) + threadIdx.z*(BLOCK_X+SMEM_PAD)*BLOCK_Y;
    if (i<NX && j<NY && k>=kstart && k<=kend) {
        u0 = d_u1[idx];
        smem[sidx] = u0;
    }
    this_thread_block().sync();
    if (i>=STENCIL_DEPTH && i<NX-STENCIL_DEPTH &&
        j>=STENCIL_DEPTH && j<NY-STENCIL_DEPTH &&
        k>=kstart+STENCIL_DEPTH && k<=kend-STENCIL_DEPTH) 
    {
        d_u2[idx] = smem_stencil(smem, d_u1, sidx, idx, u) / STENCIL_COEFF - u0;
    }
}

__global__ void gpu_stencil_smem_2d(float* __restrict__ d_u1,
			            float* __restrict__ d_u2,
                                    unsigned int jstart,
                                    unsigned int jend)
{
    float u = 0.0f, u0;
    unsigned int   i, j, idx, sidx;
    extern __shared__ float smem[];
    i  = threadIdx.x + blockIdx.x*BLOCK_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;
    idx = i + j*NX;
    sidx = threadIdx.x + threadIdx.y*(BLOCK_X+SMEM_PAD);
    if (i<NX && j>=jstart && j<=jend)
    {
        u0 = d_u1[idx];
        smem[sidx] = u0;
    }
    this_thread_block().sync();
    if (i>=STENCIL_DEPTH && i<NX-STENCIL_DEPTH &&
        j>=jstart+STENCIL_DEPTH && j<=jend-STENCIL_DEPTH) 
    {
        d_u2[idx] = smem_stencil(smem, d_u1, sidx, idx, u) / STENCIL_COEFF - u0;
    }
}

__global__ void gpu_stencil_smem_1d(float* __restrict__ d_u1,
			            float* __restrict__ d_u2,
                                    unsigned int istart,
                                    unsigned int iend)
{
    float u = 0.0f, u0;
    unsigned int i;
    extern __shared__ float smem[];
    i  = threadIdx.x + blockIdx.x*BLOCK_X;
    if (i>=istart && i<=iend)
    {
        u0 = d_u1[i];
        smem[threadIdx.x] = u0;
    }
    this_thread_block().sync();
    if (i>=STENCIL_DEPTH && i<NX-STENCIL_DEPTH)
    {
        d_u2[i] = smem_stencil(smem, d_u1, threadIdx.x, i, u) / STENCIL_COEFF - u0;
    }
}
