#include "hip/hip_runtime.h"
#include "../include/constants.h"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

__device__ float smem_stencil(float* smem, float* d_u1, unsigned int sidx, unsigned int idx, float u) {
    unsigned int d;
#pragma unroll
    for (d=STENCIL_DEPTH; d>=1; d--) {
        if (threadIdx.x >= d)        u += smem[sidx-d];
        else                         u += d_u1[idx-d];
    }
#pragma unroll
    for (d=1; d<=STENCIL_DEPTH; d++) {
        if (threadIdx.x+d < BLOCK_X) u += smem[sidx+d];
        else                         u += d_u1[idx+d];
    }
#if DIMENSIONS>1
#pragma unroll
    for (d=STENCIL_DEPTH; d>=1; d--) {
        if (threadIdx.y >= d)        u += smem[sidx-d*SMEM_X];
        else                         u += d_u1[idx-d*NX];
    }
#pragma unroll
    for (d=1; d<=STENCIL_DEPTH; d++) {
        if (threadIdx.y+d < BLOCK_Y) u += smem[sidx+d*SMEM_X];
        else                         u += d_u1[idx+d*NX];
    }
#endif
#if DIMENSIONS>2
#pragma unroll
    for (d=STENCIL_DEPTH; d>=1; d--) {
        if (threadIdx.z >= d)        u += smem[sidx-d*SMEM_X*BLOCK_Y];
        else                         u += d_u1[idx-d*NX*NY];
    }
#pragma unroll
    for (d=1; d<=STENCIL_DEPTH; d++) {
        if (threadIdx.z+d < BLOCK_Z) u += smem[sidx+d*SMEM_X*BLOCK_Y];
        else                         u += d_u1[idx+d*NX*NY];
    }
#endif
    return u;
}


__device__ inline void accumulate (float *src, unsigned int idx, float* u, int offset) {
#pragma unroll
    for (unsigned int d=1; d<=STENCIL_DEPTH; d++) *u += src[idx+d*offset];
}

// This kernel only uses shared memory for stencil dimensions smaller than the block dimensions
__device__ inline void smem_stencil_new(float* smem, float* d_u1, unsigned int sidx, unsigned int idx, float* u) {
    if (threadIdx.x >= STENCIL_DEPTH)        accumulate(smem, sidx, u, -1);
    else                                     accumulate(d_u1, idx, u, -1);

    if (threadIdx.x+STENCIL_DEPTH < BLOCK_X) accumulate(smem, sidx, u, 1);
    else                                     accumulate(d_u1, idx, u, 1);

#if DIMENSIONS>1
    if (threadIdx.y >= STENCIL_DEPTH)        accumulate(smem, sidx, u, -SMEM_X);
    else                                     accumulate(d_u1, idx, u, -NX);

    if (threadIdx.y+STENCIL_DEPTH < BLOCK_Y) accumulate(smem, sidx, u, SMEM_X);
    else                                     accumulate(d_u1, idx, u, NX);
#endif

#if DIMENSIONS>2
    if (threadIdx.z >= STENCIL_DEPTH)        accumulate(smem, sidx, u, -(SMEM_X*BLOCK_Y));
    else                                     accumulate(d_u1, idx, u, -(NX*NY));

    if (threadIdx.z+STENCIL_DEPTH < BLOCK_Z) accumulate(smem, sidx, u, (SMEM_X)*BLOCK_Y);
    else                                     accumulate(d_u1, idx, u, NX*NY);
#endif
}

__global__ void gpu_stencil_smem_3d(float* __restrict__ d_u1,
			            float* __restrict__ d_u2,
                                    unsigned int kstart,
                                    unsigned int kend)
{
    float u = 0.0f, u0;
    unsigned int   i, j, k, idx, sidx;
    extern __shared__ float smem[];
    i  = threadIdx.x + blockIdx.x*BLOCK_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;
    k  = threadIdx.z + blockIdx.z*BLOCK_Z;
    idx = i + j*NX + k*NX*NY;
    sidx = threadIdx.x + threadIdx.y*(BLOCK_X+SMEM_PAD) + threadIdx.z*(BLOCK_X+SMEM_PAD)*BLOCK_Y;
    if (i<NX && j<NY && k>=kstart && k<=kend) {
        u0 = d_u1[idx];
        smem[sidx] = u0;
    }
    this_thread_block().sync();
    if (i>=STENCIL_DEPTH && i<NX-STENCIL_DEPTH &&
        j>=STENCIL_DEPTH && j<NY-STENCIL_DEPTH &&
        k>=kstart+STENCIL_DEPTH && k<=kend-STENCIL_DEPTH) 
    {
        d_u2[idx] = smem_stencil(smem, d_u1, sidx, idx, u) / STENCIL_COEFF - u0;
    }
}

__global__ void gpu_stencil_smem_2d_unrolled(float* __restrict__ d_u1,
			                     float* __restrict__ d_u2,
                                             unsigned int jstart,
                                             unsigned int jend)
{
    unsigned int i, j, s, idx, ioff, d;
    float u;
    i  = threadIdx.x + blockIdx.x*BLOCK_X*UNROLL_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;
    __shared__ float smem[BLOCK_Y][SMEM_X];
#pragma unroll
    for (s=0; s<UNROLL_X; s++) {
        ioff = s*BLOCK_X;
        idx = (i+ioff) + j*NX;
        if ((i+ioff)<NX && j<=NY)
        {
            smem[threadIdx.y][threadIdx.x+ioff] = d_u1[idx];
        }
    }
    this_thread_block().sync();
#pragma unroll
    for (s=0; s<UNROLL_X; s++) {
        ioff = s*BLOCK_X;
        idx = (i+ioff) + j*NX;
        if ((i+ioff)>=STENCIL_DEPTH && (i+ioff)<NX-STENCIL_DEPTH &&
            j>=STENCIL_DEPTH && j<NY-STENCIL_DEPTH)
        {
            u = 0.0f;
            if (s>0) {
#pragma unroll
                for (d=1; d<=STENCIL_DEPTH; d++)
                    u += smem[threadIdx.y][threadIdx.x+ioff-d];
            } else {
#pragma unroll
                for (d=1; d<=STENCIL_DEPTH; d++) {
                    if (threadIdx.x+ioff >= d)        u += smem[threadIdx.y][threadIdx.x+ioff-d];
                    else                              u += d_u1[idx-d];
                }
            }
            if (s<UNROLL_X-1){
#pragma unroll
                for (d=1; d<=STENCIL_DEPTH; d++)
                    u += smem[threadIdx.y][threadIdx.x+ioff+d];
            } else {
#pragma unroll
                for (d=1; d<=STENCIL_DEPTH; d++) {
                    if (threadIdx.x+ioff+d < BLOCK_X) u += smem[threadIdx.y][threadIdx.x+ioff+d];
                    else                              u += d_u1[idx+d];
                }
            }

#pragma unroll
            for (d=1; d<=STENCIL_DEPTH; d++) {
                if (threadIdx.y >= d)             u += smem[threadIdx.y-d][threadIdx.x+ioff];
                else                              u += d_u1[idx-d*NX];
            }

#pragma unroll
            for (d=1; d<=STENCIL_DEPTH; d++) {
                if (threadIdx.y+d < BLOCK_Y)      u += smem[threadIdx.y+d][threadIdx.x+ioff];
                else                              u += d_u1[idx+d*NX];
            }
            d_u2[idx] = u / STENCIL_COEFF - smem[threadIdx.y][threadIdx.x+ioff];
        }
    }
}


__global__ void gpu_stencil_smem_2d_unrolled_prefetch(float* __restrict__ d_u1,
                                                      float* __restrict__ d_u2,
                                                      unsigned int jstart,
                                                      unsigned int jend)
{
    unsigned int i, j, s, idx, sidx, ioff, d, si, sj;
    float u;
    i  = threadIdx.x + blockIdx.x*BLOCK_X*UNROLL_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;
    //__shared__ float smem[SMEM_P_Y][SMEM_P_X];
    extern __shared__ float smem[];
    si = threadIdx.x + STENCIL_DEPTH;
    sj = threadIdx.y + STENCIL_DEPTH;
#pragma unroll
    for (s=0; s<UNROLL_X; s++) {
        ioff = s*BLOCK_X;
        idx = (i+ioff) + j*NX;
        sidx = si+ioff + sj*SMEM_P_X;
        if ((i+ioff)<NX && j<=NY)
        {
            if (s==0 && threadIdx.x < STENCIL_DEPTH && i >= STENCIL_DEPTH)
            {
                smem[sidx-STENCIL_DEPTH] = d_u1[idx-STENCIL_DEPTH];
            }
            if (s==UNROLL_X-1 && threadIdx.x >= BLOCK_X-STENCIL_DEPTH && i < NX-STENCIL_DEPTH)
            {
                smem[sidx+STENCIL_DEPTH] = d_u1[idx+STENCIL_DEPTH];
            }
            if (threadIdx.y < STENCIL_DEPTH && j >= STENCIL_DEPTH)
            {
                smem[sidx-STENCIL_DEPTH*SMEM_P_X] = d_u1[idx-STENCIL_DEPTH*NX];
            }
            if (threadIdx.y >= BLOCK_Y-STENCIL_DEPTH && j < NY-STENCIL_DEPTH)
            {
                smem[sidx+STENCIL_DEPTH*SMEM_P_X] = d_u1[idx+STENCIL_DEPTH*NX];
            }
            smem[sidx] = d_u1[idx];
        }
    }
    this_thread_block().sync();
#pragma unroll
    for (s=0; s<UNROLL_X; s++) {
        ioff = s*BLOCK_X;
        idx = (i+ioff) + j*NX;
        sidx = (si+ioff) + sj*SMEM_P_X;
        if ((i+ioff)>=STENCIL_DEPTH && (i+ioff)<NX-STENCIL_DEPTH &&
            j>=STENCIL_DEPTH && j<NY-STENCIL_DEPTH)
        {
            u = 0.0f;
#pragma unroll
            for (d=1; d<=STENCIL_DEPTH; d++)
            {
                u += smem[sidx-d]
                   + smem[sidx+d]
                   + smem[sidx-d*SMEM_P_X]
                   + smem[sidx+d*SMEM_P_X];
            }
            d_u2[idx] = u / STENCIL_COEFF - smem[sidx];
        }
    }
}


__global__ void gpu_stencil_smem_2d(float* __restrict__ d_u1,
			            float* __restrict__ d_u2,
                                    unsigned int jstart,
                                    unsigned int jend)
{
    float u = 0.0f, u0;
    unsigned int   i, j, idx, sidx;
    extern __shared__ float smem[];
    i  = threadIdx.x + blockIdx.x*BLOCK_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;
    idx = i + j*NX;
    sidx = threadIdx.x + threadIdx.y*(BLOCK_X+SMEM_PAD);
    if (i<NX && j>=jstart && j<=jend)
    {
        u0 = d_u1[idx];
        smem[sidx] = u0;
    }
    this_thread_block().sync();
    if (i>=STENCIL_DEPTH && i<NX-STENCIL_DEPTH &&
        j>=jstart+STENCIL_DEPTH && j<=jend-STENCIL_DEPTH) 
    {
        //d_u2[idx] = smem_stencil(smem, d_u1, sidx, idx, u) / STENCIL_COEFF - u0;
        smem_stencil_new(smem, d_u1, sidx, idx, &u);
        d_u2[idx] = u / STENCIL_COEFF - u0;
    }
}

__global__ void gpu_stencil_smem_1d(float* __restrict__ d_u1,
			            float* __restrict__ d_u2,
                                    unsigned int istart,
                                    unsigned int iend)
{
    float u = 0.0f, u0;
    unsigned int i;
    extern __shared__ float smem[];
    i  = threadIdx.x + blockIdx.x*BLOCK_X;
    if (i>=istart && i<=iend)
    {
        u0 = d_u1[i];
        smem[threadIdx.x] = u0;
    }
    this_thread_block().sync();
    if (i>=STENCIL_DEPTH && i<NX-STENCIL_DEPTH)
    {
        d_u2[i] = smem_stencil(smem, d_u1, threadIdx.x, i, u) / STENCIL_COEFF - u0;
    }
}
