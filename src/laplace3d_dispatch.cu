#include "../include/constants.h"
#include "../include/hip/hip_runtime_api.h"
#include "laplace3d_kernel.cu"

void dispatch_kernels(float *d_u1, float *d_u2) {
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    dim3 grid(1 + (NX-1)/BLOCK_X, 1 + (NY-1)/BLOCK_Y, 1 + (NZ-1)/BLOCK_Z);
    float *d_tmp;
    for (int i=0; i<ITERATIONS/SMEM_HALO_DEPTH; i++) {
        if (SMEM) gpu_laplace3d_smem<<<grid, block>>>(d_u1, d_u2, 0, NY-1);
        else      gpu_laplace3d_base<<<grid, block>>>(d_u1, d_u2, 0, NY-1);
        getLastCudaError("gpu_laplace3d execution failed\n");
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
    }
}

void dispatch_cooperative_groups_kernels(float *d_u1, float *d_u2) {
    int device = 0;
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    int numBlocksPerSm = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    if (SMEM)
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm,
                                                      (void*)gpu_laplace3d_coop_smem,
                                                      BLOCK_X*BLOCK_Y*BLOCK_Z,
                                                      0);
    else
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm,
                                                      (void*)gpu_laplace3d_coop,
                                                      BLOCK_X*BLOCK_Y*BLOCK_Z,
                                                      0);
    dim3 grid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
    void *args[] = {
        &d_u1,
        &d_u2
    };
    if (SMEM)
        hipLaunchCooperativeKernel((void*)gpu_laplace3d_coop_smem,
                                    grid,
                                    block,
                                    args);
    else
        hipLaunchCooperativeKernel((void*)gpu_laplace3d_coop,
                                    grid,
                                    block,
                                    args);
    getLastCudaError("gpu_laplace3d execution failed\n");
}

void dispatch_multi_gpu_kernels(float **d_u1, float **d_u2, hipStream_t *streams) {
    dim3 block(BLOCK_X,BLOCK_Y);
    dim3 grid(1 + (NX-1)/BLOCK_X, 1 + (NY-1)/BLOCK_Y);
    float *d_tmp;
    int i, s, n;
    int jstart, jend;

    int bot = HALO_DEPTH;
    int top = HALO_DEPTH+NY/NGPUS-1;

    for (i=0; i<ITERATIONS/HALO_DEPTH; i++) {
        for (s=0; s<NGPUS; s++) {
            hipSetDevice(s);
            if (s==0)
                CU(hipMemcpyPeerAsync(d_u1[s] + (top+1) * NX,
                                       s,
                                       d_u1[s+1] + bot * NX,
                                       s+1,
                                       BORDER_BYTES,
                                       streams[s]));
            else if (s==NGPUS-1)
                CU(hipMemcpyPeerAsync(d_u1[s],
                                       s,
                                       d_u1[s-1] + top * NX,
                                       s-1,
                                       BORDER_BYTES,
                                       streams[s]));
            else {
                CU(hipMemcpyPeerAsync(d_u1[s],
                                       s,
                                       d_u1[s-1] + top * NX,
                                       s-1,
                                       BORDER_BYTES,
                                       streams[s]));
                CU(hipMemcpyPeerAsync(d_u1[s] + (top+1) * NX,
                                       s,
                                       d_u1[s+1] + bot * NX,
                                       s+1,
                                       BORDER_BYTES,
                                       streams[s]));
            }
        }
        for (n = 0; n < HALO_DEPTH; n++) {
            for (s=0; s<NGPUS; s++) {
                hipSetDevice(s);
                jstart = bot;
                jend = top;
                if (s==0) {
                    jstart = bot;
                    jend = top+HALO_DEPTH;
                } else if (s==NGPUS-1) {
                    jstart = 0;
                    jend = top;
                } else {
                    jstart = 0;
                    jend = top+HALO_DEPTH;
                }
                if (SMEM)
                    gpu_laplace3d_smem<<<grid, block, 0, streams[s]>>>(d_u1[s],
                                                                       d_u2[s],
                                                                       jstart,
                                                                       jend);
                else
                    gpu_laplace3d_base<<<grid, block, 0, streams[s]>>>(d_u1[s],
                                                                       d_u2[s],
                                                                       jstart,
                                                                       jend);
                getLastCudaError("gpu_laplace3d execution failed\n");
            }
            for (s=0; s<NGPUS; s++) {
                hipSetDevice(s);
                hipStreamSynchronize(streams[s]);
                d_tmp = d_u1[s]; d_u1[s] = d_u2[s]; d_u2[s] = d_tmp; // swap d_u1 and d_u2
            }
        }
    }
}
