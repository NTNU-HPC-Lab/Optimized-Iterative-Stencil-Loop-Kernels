#include "../include/constants.h"
#include "../include/hip/hip_runtime_api.h"
#include "stencil_kernel_base.cu"
#include "stencil_kernel_smem.cu"
#include "stencil_kernel_coop.cu"

typedef void (*kernel)      (float*,float*,unsigned int,unsigned int);
typedef void (*coop_kernel) (float*,float*);

kernel      get_kernel()      { 
    if (DIMENSIONS==3) {
        if (SMEM)       return gpu_stencil_smem_3d;
        if (UNROLL_X>1) return gpu_stencil_base_3d_unrolled;
        return gpu_stencil_base_3d;
    } else if (DIMENSIONS==2) {
        if (SMEM)       return gpu_stencil_smem_2d;
        if (UNROLL_X>1) return gpu_stencil_base_2d_unrolled;
        return gpu_stencil_base_2d;
    } else {
        if (SMEM)       return gpu_stencil_smem_1d;
        if (UNROLL_X>1) return gpu_stencil_base_1d_unrolled;
        return gpu_stencil_base_1d;
    }
}

coop_kernel get_coop_kernel() { return gpu_stencil_coop; }

void dispatch_kernels(float *d_u1, float *d_u2) {
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    dim3 grid((1+(NX-1)/BLOCK_X)/UNROLL_X);
    if (DIMENSIONS>1) grid.y = 1+(NY-1)/BLOCK_Y;
    if (DIMENSIONS>2) grid.z = 1+(NZ-1)/BLOCK_Z;
    float *d_tmp;
    unsigned int smem = 0;
    if (SMEM) {
        smem = (BLOCK_X+SMEM_PAD)*BLOCK_Y*BLOCK_Z*sizeof(float);
        //hipFuncSetAttribute(get_kernel(), hipFuncAttributeMaxDynamicSharedMemorySize, smem);
        // Max on V100: hipFuncSetAttribute(reinterpret_cast<const void*>(gpu_stencil_smem), hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
    }
    for (int i=0; i<ITERATIONS; i++) {
        get_kernel()<<<grid, block, smem>>>(d_u1, d_u2, 0, NZ-1);
        getLastCudaError("kernel execution failed\n");
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
    }
}

void dispatch_cooperative_groups_kernels(float *d_u1, float *d_u2) {
    int device = 0;
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    int numBlocksPerSm = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm,
                                                  get_coop_kernel(),
                                                  BLOCK_X*BLOCK_Y*BLOCK_Z,
                                                  0);
    dim3 grid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
    void *args[] = { &d_u1, &d_u2 };
    hipLaunchCooperativeKernel((void*)get_coop_kernel(),
                                grid,
                                block,
                                args);
    getLastCudaError("kernel execution failed\n");
}

void send_upper_ghost_zone(float **d_u1, unsigned int dev, hipStream_t* streams) {
    CU(hipMemcpyPeerAsync(d_u1[dev+1],
                           dev+1,
                           d_u1[dev] + (INTERNAL_END-HALO_DEPTH) * BORDER_SIZE,
                           dev,
                           GHOST_ZONE_BYTES,
                           streams[dev]));
}

void send_lower_ghost_zone(float **d_u1, unsigned int dev, hipStream_t* streams) {
    CU(hipMemcpyPeerAsync(d_u1[dev-1] + INTERNAL_END * BORDER_SIZE,
                           dev-1,
                           d_u1[dev] + INTERNAL_START * BORDER_SIZE,
                           dev,
                           GHOST_ZONE_BYTES,
                           streams[dev]));
}

void dispatch_multi_gpu_kernels(float **d_u1, float **d_u2, hipStream_t *streams) {
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    dim3 grid(1+(NX-1)/BLOCK_X, 1+(NY-1)/BLOCK_Y, 1+(NZ-1)/BLOCK_Z);
    float **d_tmp;
    //int i, s, n, kstart, kend;
    int i, s;
    unsigned int kstart, kend;
    //for (i=0; i<ITERATIONS/HALO_DEPTH; i++) {
    for (i=0; i<ITERATIONS; i++) {
        for (s=0; s<NGPUS-1; s++) send_upper_ghost_zone(d_u1, s, streams);
        for (s=1; s<NGPUS; s++)   send_lower_ghost_zone(d_u1, s, streams);
        for (s=0; s<NGPUS; s++)   CU(hipStreamSynchronize(streams[s]));
        //for (n=0; n<HALO_DEPTH; n++) {
        for (s=0; s<NGPUS; s++) {
            CU(hipSetDevice(s));
            kstart = 0;
            kend   = INTERNAL_END-1+HALO_DEPTH;
            if      (s==0)       kstart = INTERNAL_START;
            else if (s==NGPUS-1) kend   = INTERNAL_END-1;
            unsigned int smem = 0;
            if (SMEM) {
                smem = (BLOCK_X+SMEM_PAD)*BLOCK_Y*BLOCK_Z*sizeof(float);
                hipFuncSetAttribute(get_kernel(), hipFuncAttributeMaxDynamicSharedMemorySize, smem);
            }
            get_kernel()<<<grid, block, smem, streams[s]>>>(d_u1[s], d_u2[s], kstart, kend);
            getLastCudaError("kernel execution failed\n");
        }
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
        //}
        for (s=0; s<NGPUS; s++) CU(hipStreamSynchronize(streams[s]));
    }
}
