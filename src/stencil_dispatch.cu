#include "../include/constants.h"
#include "../include/hip/hip_runtime_api.h"
#include "stencil_kernel_base.cu"
#include "stencil_kernel_smem.cu"
#include "stencil_kernel_coop.cu"

typedef void (*kernel)      (float*,float*,unsigned int,unsigned int);
typedef void (*coop_kernel) (float*,float*);


kernel get_kernel() { 
    if (DIMENSIONS==3) {
        if (SMEM)       return gpu_stencil_smem_3d;
        if (UNROLL_X>1) return gpu_stencil_base_3d_unrolled;
        return gpu_stencil_base_3d;
    } else if (DIMENSIONS==2) {
        if (SMEM) {
            if (REGISTER) return gpu_stencil_smem_2d_register;
            if (UNROLL_X>1) {
                if (PREFETCH) return gpu_stencil_smem_2d_unrolled_prefetch;
                else          return gpu_stencil_smem_2d_unrolled;
            }
            if (PREFETCH) return gpu_stencil_smem_2d_prefetch;
            else          return gpu_stencil_smem_2d;
        }
        if (UNROLL_X>1) return gpu_stencil_base_2d_unrolled;
        else            return gpu_stencil_base_2d;
    } else {
        if (SMEM)       return gpu_stencil_smem_1d;
        if (UNROLL_X>1) return gpu_stencil_base_1d_unrolled;
        return gpu_stencil_base_1d;
    }
}

coop_kernel get_coop_kernel() { return gpu_stencil_coop; }

void set_smem(unsigned int *smem) {
        if (!SMEM)        {*smem = 0; return;}
        else if (PREFETCH) *smem = SMEM_P_X*SMEM_P_Y*BLOCK_Z*sizeof(float);
        else if (REGISTER) *smem = SMEM_P_X*BLOCK_Y*BLOCK_Z*sizeof(float);
        else               *smem = SMEM_X*BLOCK_Y*BLOCK_Z*sizeof(float);
        hipFuncSetAttribute(get_kernel(), hipFuncAttributeMaxDynamicSharedMemorySize, *smem);
        // Max on V100: hipFuncSetAttribute(reinterpret_cast<const void*>(gpu_stencil_smem), hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
}

void dispatch_kernels(float *d_u1, float *d_u2) {
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    dim3 grid((1+(NX-1)/BLOCK_X)/UNROLL_X);
    if (DIMENSIONS>1) grid.y = 1+(NY-1)/BLOCK_Y;
    if (DIMENSIONS>2) grid.z = 1+(NZ-1)/BLOCK_Z;
    float *d_tmp;
    unsigned int smem;
    set_smem(&smem);
    for (int i=0; i<ITERATIONS; i++) {
        get_kernel()<<<grid, block, smem>>>(d_u1, d_u2, 0, NZ-1);
        getLastCudaError("kernel execution failed\n");
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
    }
}

void dispatch_cooperative_groups_kernels(float *d_u1, float *d_u2) {
    int device = 0;
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    int numBlocksPerSm = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm,
                                                  get_coop_kernel(),
                                                  BLOCK_X*BLOCK_Y*BLOCK_Z,
                                                  0);
    dim3 grid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
    void *args[] = { &d_u1, &d_u2 };
    hipLaunchCooperativeKernel((void*)get_coop_kernel(),
                                grid,
                                block,
                                args);
    getLastCudaError("kernel execution failed\n");
}

void send_upper_ghost_zone(float **d_u1, unsigned int dev, hipStream_t* streams) {
    CU(hipMemcpyPeerAsync(d_u1[dev+1],
                           dev+1,
                           d_u1[dev] + (INTERNAL_END-HALO_DEPTH) * BORDER_SIZE,
                           dev,
                           GHOST_ZONE_BYTES,
                           streams[dev]));
}

void send_lower_ghost_zone(float **d_u1, unsigned int dev, hipStream_t* streams) {
    CU(hipMemcpyPeerAsync(d_u1[dev-1] + INTERNAL_END * BORDER_SIZE,
                           dev-1,
                           d_u1[dev] + INTERNAL_START * BORDER_SIZE,
                           dev,
                           GHOST_ZONE_BYTES,
                           streams[dev]));
}

void dispatch_multi_gpu_kernels(float **d_u1, float **d_u2, hipStream_t *streams) {
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    dim3 grid((1+(NX-1)/BLOCK_X)/UNROLL_X);
    if      (DIMENSIONS==2) grid.y = 1+(NY/NGPUS+2*HALO_DEPTH-1)/BLOCK_Y;
    else if (DIMENSIONS==3) {
        grid.y = 1+(NY-1)/BLOCK_Y;
        grid.z = 1+(NZ/NGPUS+2*HALO_DEPTH-1)/BLOCK_Z;
    }

    float **d_tmp;
    //int i, s, n, kstart, kend;
    int i, s;
    unsigned int smem, kstart, kend;
    set_smem(&smem);
    //for (i=0; i<ITERATIONS/HALO_DEPTH; i++) {
    for (i=0; i<ITERATIONS; i++) {
        for (s=0; s<NGPUS-1; s++) send_upper_ghost_zone(d_u1, s, streams);
        for (s=1; s<NGPUS; s++)   send_lower_ghost_zone(d_u1, s, streams);
        for (s=0; s<NGPUS; s++)   CU(hipStreamSynchronize(streams[s]));
        //for (n=0; n<HALO_DEPTH; n++) {
        for (s=0; s<NGPUS; s++) {
            CU(hipSetDevice(s));
            kstart = 0;
            kend   = INTERNAL_END-1+HALO_DEPTH;
            if      (s==0)       kstart = INTERNAL_START;
            else if (s==NGPUS-1) kend   = INTERNAL_END-1;
            get_kernel()<<<grid, block, smem, streams[s]>>>(d_u1[s], d_u2[s], kstart, kend);
            getLastCudaError("kernel execution failed\n");
        }
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
        //}
        for (s=0; s<NGPUS; s++) CU(hipStreamSynchronize(streams[s]));
    }
}
