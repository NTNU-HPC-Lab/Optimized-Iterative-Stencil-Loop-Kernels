#include "../include/constants.h"
#include "../include/hip/hip_runtime_api.h"
#include "stencil_kernel.cu"

typedef void (*kernel)      (float*,float*,int,int);
typedef void (*coop_kernel) (float*,float*);

kernel      get_kernel()      { return SMEM ? gpu_stencil_smem      : gpu_stencil_base; }
coop_kernel get_coop_kernel() { return SMEM ? gpu_stencil_coop_smem : gpu_stencil_coop; }

void dispatch_kernels(float *d_u1, float *d_u2) {
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    dim3 grid(1+(NX-1)/BLOCK_X, 1+(NY-1)/BLOCK_Y, 1+(NZ-1)/BLOCK_Z);
    float *d_tmp;
    unsigned int smem = 0;
    if (SMEM) {
        smem = BLOCK_X*BLOCK_Y*BLOCK_Z*sizeof(float);
        hipFuncSetAttribute(reinterpret_cast<const void*>(gpu_stencil_smem), hipFuncAttributeMaxDynamicSharedMemorySize, smem);
    }
    for (int i=0; i<ITERATIONS; i++) {
        get_kernel()<<<grid, block, smem>>>(d_u1, d_u2, 0, NZ-1);
        getLastCudaError("kernel execution failed\n");
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
    }
}

void dispatch_cooperative_groups_kernels(float *d_u1, float *d_u2) {
    int device = 0;
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    int numBlocksPerSm = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm,
                                                  get_coop_kernel(),
                                                  BLOCK_X*BLOCK_Y*BLOCK_Z,
                                                  0);
    dim3 grid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
    void *args[] = { &d_u1, &d_u2 };
    hipLaunchCooperativeKernel((void*)get_coop_kernel(),
                                grid,
                                block,
                                args);
    getLastCudaError("kernel execution failed\n");
}

void send_upper_ghost_zone(float **d_u1, unsigned int dev, hipStream_t* streams) {
    CU(hipMemcpyPeerAsync(d_u1[dev+1],
                           dev+1,
                           d_u1[dev] + (INTERNAL_END-HALO_DEPTH) * BORDER_SIZE,
                           dev,
                           GHOST_ZONE_BYTES,
                           streams[dev]));
}

void send_lower_ghost_zone(float **d_u1, unsigned int dev, hipStream_t* streams) {
    CU(hipMemcpyPeerAsync(d_u1[dev-1] + INTERNAL_END * BORDER_SIZE,
                           dev-1,
                           d_u1[dev] + INTERNAL_START * BORDER_SIZE,
                           dev,
                           GHOST_ZONE_BYTES,
                           streams[dev]));
}

void dispatch_multi_gpu_kernels(float **d_u1, float **d_u2, hipStream_t *streams) {
    dim3 block(BLOCK_X,BLOCK_Y,BLOCK_Z);
    dim3 grid(1+(NX-1)/BLOCK_X, 1+(NY-1)/BLOCK_Y, 1+(NZ-1)/BLOCK_Z);
    float **d_tmp;
    //int i, s, n, kstart, kend;
    int i, s, kstart, kend;
    //for (i=0; i<ITERATIONS/HALO_DEPTH; i++) {
    for (i=0; i<ITERATIONS; i++) {
        for (s=0; s<NGPUS-1; s++) send_upper_ghost_zone(d_u1, s, streams);
        for (s=1; s<NGPUS; s++)   send_lower_ghost_zone(d_u1, s, streams);
        for (s=0; s<NGPUS; s++)   CU(hipStreamSynchronize(streams[s]));
        //for (n=0; n<HALO_DEPTH; n++) {
        for (s=0; s<NGPUS; s++) {
            CU(hipSetDevice(s));
            kstart = 0;
            kend   = INTERNAL_END-1+HALO_DEPTH;
            if      (s==0)       kstart = INTERNAL_START;
            else if (s==NGPUS-1) kend   = INTERNAL_END-1;
            unsigned int smem = 0;
            if (SMEM) {
                smem = BLOCK_X*BLOCK_Y*BLOCK_Z*sizeof(float);
                hipFuncSetAttribute(reinterpret_cast<const void*>(gpu_stencil_smem), hipFuncAttributeMaxDynamicSharedMemorySize, smem);
            }
            get_kernel()<<<grid, block, smem, streams[s]>>>(d_u1[s], d_u2[s], kstart, kend);
            getLastCudaError("kernel execution failed\n");
        }
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
        //}
        for (s=0; s<NGPUS; s++) CU(hipStreamSynchronize(streams[s]));
    }
}
