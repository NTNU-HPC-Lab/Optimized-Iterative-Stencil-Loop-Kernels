#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_kernel.cu"


void dispatch_kernels(float **d_u1, float **d_u2, hipStream_t *streams) {
    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(1 + (NX-1)/BLOCK_X, 1 + (NY-1)/BLOCK_Y);

    float *d_tmp;
    int i, s;
    for (i=0; i<ITERATIONS; i++) {
        CU(hipMemcpyAsync(d_u1[1], d_u1[0] + (NY/NGPUS) * NX,
                        NX*sizeof(float), hipMemcpyDefault, streams[0]));
        CU(hipMemcpyAsync(d_u1[0] + (NY/NGPUS + 1) * NX, d_u1[1] + NX,
                        NX*sizeof(float), hipMemcpyDefault, streams[0]));
        hipSetDevice(0);
        gpu_laplace2d_base<<<dimGrid, dimBlock, 0, streams[0]>>>(d_u1[0], d_u2[0], 1, NY/NGPUS+1);
        getLastCudaError("gpu_laplace2d (dev 0) execution failed\n");

        hipSetDevice(1);
        gpu_laplace2d_base<<<dimGrid, dimBlock, 0, streams[1]>>>(d_u1[1], d_u2[1], 0, NY/NGPUS);
        getLastCudaError("gpu_laplace2d (dev 1) execution failed\n");

        //if (SMEM) gpu_laplace2d_smem<<<dimGrid, dimBlock, 0, streams[s]>>>(d_u1, d_u2, start, end);

        //CU(hipMemcpyAsync(d_u1[1], d_u1[0] + (NY/NGPUS) * NX,
        //                NX*sizeof(float), hipMemcpyDefault, streams[1]));
        //CU(hipMemcpyAsync(d_u1[0] + (NY/NGPUS + 1) * NX, d_u1[1] + NX,
        //                NX*sizeof(float), hipMemcpyDefault, streams[0]));

        //hipSetDevice(0);
        //gpu_laplace2d_base<<<dimGrid, dimBlock, 0, streams[0]>>>(d_u1[s], d_u2[s], 1, NY/NGPUS);
        //getLastCudaError("gpu_laplace2d execution failed\n");
        //hipSetDevice(1);
        //gpu_laplace2d_base<<<dimGrid, dimBlock, 0, streams[1]>>>(d_u1[s], d_u2[s], 0, NY/NGPUS);
        //getLastCudaError("gpu_laplace2d execution failed\n");

        //for (s=0; s<NGPUS; s++) {
            //if (SMEM) gpu_laplace2d_smem<<<dimGrid, dimBlock, 0, streams[s]>>>(d_u1, d_u2, start, end);
        //}
        //CU(hipMemcpyAsync(d_u1[0] + dst_skip[1], d_u1[1] + src_skip[1],
        //            iexchange, hipMemcpyDefault, stream_halo[1]));
        // TODO: Communicate border
        for (s=0; s<NGPUS; s++) {
            hipSetDevice(s);
            hipStreamSynchronize(streams[s]);
            d_tmp = d_u1[s]; d_u1[s] = d_u2[s]; d_u2[s] = d_tmp; // swap d_u1 and d_u2
        }
        // TODO (multi-gpu): Exchange borders before continuing
    }
}

void dispatch_cooperative_groups_kernels(float *d_u1, float *d_u2) {
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(deviceProp.multiProcessorCount, 1);
    void *args[] = {
        &d_u1,
        &d_u2
    };
    if (SMEM) hipLaunchCooperativeKernel((void*)gpu_laplace2d_coop_smem, dimGrid, dimBlock, args);
    else hipLaunchCooperativeKernel((void*)gpu_laplace2d_coop, dimGrid, dimBlock, args);
    getLastCudaError("gpu_laplace2d execution failed\n");
}
