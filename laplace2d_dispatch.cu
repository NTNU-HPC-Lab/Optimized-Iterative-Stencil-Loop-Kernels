#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_kernel.cu"


void dispatch_kernels(float **d_u1, float **d_u2, hipStream_t *streams) {
    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(1 + (NX-1)/BLOCK_X, 1 + (NY-1)/BLOCK_Y);

    float *d_tmp;
    int i, s;
    for (i=0; i<ITERATIONS; i++) {
        for (s=0; s<NGPUS; s++) {
            if (s==0)
                CU(hipMemcpyAsync(d_u1[s+1], d_u1[s] + (NY/NGPUS) * NX,
                                   NX*sizeof(float), hipMemcpyDefault, streams[s+1]));
            else if (s==NGPUS-1)
                CU(hipMemcpyAsync(d_u1[s-1] + (NY/NGPUS + 1) * NX, d_u1[s] + NX,
                                   NX*sizeof(float), hipMemcpyDefault, streams[s-1]));
            else {
                CU(hipMemcpyAsync(d_u1[s+1], d_u1[s] + (NY/NGPUS) * NX,
                                   NX*sizeof(float), hipMemcpyDefault, streams[s+1]));
                CU(hipMemcpyAsync(d_u1[s-1] + (NY/NGPUS + 1) * NX, d_u1[s] + NX,
                                   NX*sizeof(float), hipMemcpyDefault, streams[s-1]));
            }
        }
        int jstart, jend;
        for (s=0; s<NGPUS; s++) {
            hipSetDevice(s);
            if (s==0) {
                jstart = 1;
                jend = NY/NGPUS+1;
            } else if (s==NGPUS-1) {
                jstart = 0;
                jend = NY/NGPUS;
            } else {
                jstart = 0;
                jend = NY/NGPUS+1;
            }
            if (SMEM) gpu_laplace2d_smem<<<dimGrid, dimBlock, 0, streams[s]>>>(d_u1[s], d_u2[s], jstart, jend);
            else      gpu_laplace2d_base<<<dimGrid, dimBlock, 0, streams[s]>>>(d_u1[s], d_u2[s], jstart, jend);
            getLastCudaError("gpu_laplace2d execution failed\n");
        }

        //for (s=0; s<NGPUS; s++) {
            //if (SMEM) gpu_laplace2d_smem<<<dimGrid, dimBlock, 0, streams[s]>>>(d_u1, d_u2, start, end);
        //}
        
        for (s=0; s<NGPUS; s++) {
            hipSetDevice(s);
            hipStreamSynchronize(streams[s]);
            d_tmp = d_u1[s]; d_u1[s] = d_u2[s]; d_u2[s] = d_tmp; // swap d_u1 and d_u2
        }
    }
}

void dispatch_cooperative_groups_kernels(float *d_u1, float *d_u2) {
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(deviceProp.multiProcessorCount, 1);
    void *args[] = {
        &d_u1,
        &d_u2
    };
    if (SMEM) hipLaunchCooperativeKernel((void*)gpu_laplace2d_coop_smem, dimGrid, dimBlock, args);
    else hipLaunchCooperativeKernel((void*)gpu_laplace2d_coop, dimGrid, dimBlock, args);
    getLastCudaError("gpu_laplace2d execution failed\n");
}
