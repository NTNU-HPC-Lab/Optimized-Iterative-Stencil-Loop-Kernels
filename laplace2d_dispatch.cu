#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_kernel.cu"

void dispatch_kernels(float *d_u1, float *d_u2) {
    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(1 + (NX-1)/BLOCK_X, 1 + (NY-1)/BLOCK_Y);
    float *d_tmp;
    for (int i=0; i<ITERATIONS; i++) {
        if (SMEM) gpu_laplace2d_smem<<<dimGrid, dimBlock>>>(d_u1, d_u2, 0, NY-1);
        else      gpu_laplace2d_base<<<dimGrid, dimBlock>>>(d_u1, d_u2, 0, NY-1);
        getLastCudaError("gpu_laplace2d execution failed\n");
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
    }
}

void dispatch_cooperative_groups_kernels(float *d_u1, float *d_u2) {
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(deviceProp.multiProcessorCount, 1);
    void *args[] = {
        &d_u1,
        &d_u2
    };
    if (SMEM) hipLaunchCooperativeKernel((void*)gpu_laplace2d_coop_smem, dimGrid, dimBlock, args);
    else hipLaunchCooperativeKernel((void*)gpu_laplace2d_coop, dimGrid, dimBlock, args);
    getLastCudaError("gpu_laplace2d execution failed\n");
}

void dispatch_multi_gpu_kernels(float **d_u1, float **d_u2) {
    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(1 + (NX-1)/BLOCK_X, 1 + (NY-1)/BLOCK_Y);
    float *d_tmp;
    int i, s;
    for (i=0; i<ITERATIONS; i++) {
        for (s=0; s<NGPUS; s++) {
            hipSetDevice(s);
            if (s==0)
                CU(hipMemcpy(d_u1[s] + (NY/NGPUS + 1) * NX, d_u1[s+1] + NX,
                                   NX*sizeof(float), hipMemcpyDeviceToDevice));
            else if (s==NGPUS-1)
                CU(hipMemcpy(d_u1[s], d_u1[s-1] + (NY/NGPUS) * NX,
                                   NX*sizeof(float), hipMemcpyDeviceToDevice));
            else {
                CU(hipMemcpy(d_u1[s], d_u1[s-1] + (NY/NGPUS) * NX,
                                   NX*sizeof(float), hipMemcpyDeviceToDevice));
                CU(hipMemcpy(d_u1[s] + (NY/NGPUS + 1) * NX, d_u1[s+1] + NX,
                                   NX*sizeof(float), hipMemcpyDeviceToDevice));
            }
            //if (s==0)
            //    CU(hipMemcpy(d_u1[s+1], d_u1[s] + (NY/NGPUS) * NX,
            //                       NX*sizeof(float), hipMemcpyDeviceToDevice));
            //else if (s==NGPUS-1)
            //    CU(hipMemcpy(d_u1[s-1] + (NY/NGPUS + 1) * NX, d_u1[s] + NX,
            //                       NX*sizeof(float), hipMemcpyDeviceToDevice));
            //else {
            //    CU(hipMemcpy(d_u1[s+1], d_u1[s] + (NY/NGPUS) * NX,
            //                       NX*sizeof(float), hipMemcpyDeviceToDevice));
            //    CU(hipMemcpy(d_u1[s-1] + (NY/NGPUS + 1) * NX, d_u1[s] + NX,
            //                       NX*sizeof(float), hipMemcpyDeviceToDevice));
            //}
        }
        int jstart, jend;
        for (s=0; s<NGPUS; s++) {
            hipSetDevice(s);
            if (s==0) {
                jstart = 1;
                jend = NY/NGPUS+1;
            } else if (s==NGPUS-1) {
                jstart = 0;
                jend = NY/NGPUS;
            } else {
                jstart = 0;
                jend = NY/NGPUS+1;
            }
            if (SMEM) gpu_laplace2d_smem<<<dimGrid, dimBlock>>>(d_u1[s], d_u2[s], jstart, jend);
            else      gpu_laplace2d_base<<<dimGrid, dimBlock>>>(d_u1[s], d_u2[s], jstart, jend);
            getLastCudaError("gpu_laplace2d execution failed\n");
        }

        for (s=0; s<NGPUS; s++) {
            hipSetDevice(s);
            hipDeviceSynchronize();
            d_tmp = d_u1[s]; d_u1[s] = d_u2[s]; d_u2[s] = d_tmp; // swap d_u1 and d_u2
        }
    }
}
