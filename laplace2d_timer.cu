
#include <hip/hip_runtime.h>
#include <stdio.h>

void stop_timer(hipEvent_t *start, hipEvent_t *stop, float *m, const char * format)
{
    hipEventRecord(*stop);
    hipEventSynchronize(*stop);
    hipEventElapsedTime(m, *start, *stop);
    printf(format, *m);
}
