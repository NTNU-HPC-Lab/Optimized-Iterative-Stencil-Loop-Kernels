
#include <hip/hip_runtime.h>
//
// Notes: one thread per node in the 3D block
//

// device code
//

__global__ void GPU_laplace3d(const float* __restrict__ d_u1,
			      float* __restrict__ d_u2,
                              const int blockx,
                              const int blocky,
                              const int blockz,
                              const int nx,
                              const int ny,
                              const int nz)
{
  int   i, j, k, indg, ioff, joff, koff;
  float u2, sixth=1.0f/6.0f;

  //
  // define global indices and array offsets
  //

  i    = threadIdx.x + blockIdx.x*blockx;
  j    = threadIdx.y + blockIdx.y*blocky;
  k    = threadIdx.z + blockIdx.z*blockz;

  ioff = 1;
  joff = nx;
  koff = nx*ny;

  indg = i + j*joff + k*koff;

  if (i>=0 && i<=nx-1 && j>=0 && j<=ny-1 && k>=0 && k<=nz-1) {
    if (i==0 || i==nx-1 || j==0 || j==ny-1 || k==0 || k==nz-1) {
      u2 = d_u1[indg];  // Dirichlet b.c.'s
    }
    else {
      float ival[] ={
        d_u1[indg-ioff],
        d_u1[indg+ioff]
      };
      float jval[] ={
        d_u1[indg-joff],
        d_u1[indg+joff]
      };
      float kval[] ={
        d_u1[indg-koff],
        d_u1[indg+koff]
      };
      float tmp = 0.0f;
      for (int d=0; d<2; d++) tmp += ival[d] + jval[d] + kval[d];
      u2 = tmp * sixth;
    }
    d_u2[indg] = u2;
  }
}
