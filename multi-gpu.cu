#include "hip/hip_runtime.h"
#include "stdio.h"
#include "common/common.h"

__global__ void printGPU(int dev)
{
    printf("Hello from GPU %d!\n", dev);
}

void printSmCount()
{
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Number of SMs: %d\n", deviceProp.multiProcessorCount);
}

void printComputeCapabilities(int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printf("Device %d has compute capability %d.%d.\n", i, devProp.major, devProp.minor); // 7.5
    }
}


int main(int argc, char *argv[])
{
    // Fetch number of GPUs
    int ngpus; // 2
    hipGetDeviceCount(&ngpus);

    // Print GPU properties
    printSmCount();
    printComputeCapabilities(ngpus);

    // Create streams (one per GPU)
    hipStream_t streams[ngpus];
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        hipStreamCreate(&streams[i]);
    }

    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        printGPU<<<1, 1>>>(i);
        hipDeviceSynchronize();
    }
    
    // Destroy streams
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        hipStreamDestroy(streams[i]);
    }
}
