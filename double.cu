#include "hip/hip_runtime.h"
#include "common.h"
#include "utils.h"
#include <stdio.h>
#include "hip/hip_cooperative_groups.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace cooperative_groups;

__global__
void test(int *d1, int *d2, int device)
{
    multi_grid_group mgg = this_multi_grid();
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    d1[idx] = idx * device;
    mgg.sync();
    memcpy(d2, d1, 5 * sizeof(int));//, hipMemcpyDeviceToDevice);
    // Race-condition faktisk...
}

int main(int argc, char *argv[]) {

    int ngpus = 4;

    ENABLE_P2P(ngpus);

    hipLaunchParams *launchParams = (hipLaunchParams*) malloc(sizeof(hipLaunchParams) * ngpus);
    hipStream_t     *streams      = (hipStream_t*)     malloc(sizeof(hipStream_t)     * ngpus);

    // set up gpu card
    int *d_u[ngpus];

    size_t isize = 5;
    size_t ibyte = isize * sizeof(int);

    int *host_ref;
    hipHostMalloc((void **) &host_ref, ibyte * ngpus);

    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipStreamCreate(&streams[i]));
        //CHECK(hipMemcpyToSymbol(HIP_SYMBOL(device), &i, sizeof(int)));
        //int d;
        //hipGetDevice(&d);
        //CHECK(hipMemcpyToSymbol(HIP_SYMBOL(device), &i, sizeof(int)));
        //CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(device), &i, sizeof(int), 0, hipMemcpyHostToDevice, streams[i]));
    }



    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipMalloc((void **) &d_u[i], ibyte));
    }

    CHECK(hipSetDevice(0));
    dim3 block(isize);
    dim3 grid(1);


    void *args[ngpus][3];
    int devices[ngpus];

    for (int i = 0; i < 4; i++)
    {
        devices[i] = i;
    }

    for (int i = 0; i < ngpus; i++)
    {
        args[i][0] = &d_u[i];
        args[i][1] = &d_u[(i+1)%ngpus];
        args[i][2] = &devices[i];
        launchParams[i].func = (void*)test;
        launchParams[i].gridDim = grid;
        launchParams[i].blockDim = block;
        launchParams[i].sharedMem = 0;
        launchParams[i].stream = streams[i];
        launchParams[i].args = args[i];
    }

    hipLaunchCooperativeKernelMultiDevice(launchParams, ngpus);

    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipMemcpyAsync(&host_ref[i * isize], d_u[i], ibyte, hipMemcpyDeviceToHost, streams[i]));
    }

    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipStreamDestroy(streams[i]));
        CHECK(hipFree(d_u[i]));
    }

    for (int d = 0; d < ngpus; d++) {
        for (int h = 0; h < isize; h++) {
            printf("%d ", host_ref[h]);
        }
        printf("\n");
    }

    CHECK(hipDeviceReset());

    hipFree(host_ref);
    free(launchParams);
    free(streams);
}
