#include "hip/hip_runtime.h"
#include "common.h"
#include "utils.h"
#include <stdio.h>
#include "hip/hip_cooperative_groups.h"

__global__
void test(int *u)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //u[idx] = 1;
}

int main(int argc, char *argv[]) {

    int ngpus = 2;

    hipLaunchParams *launchParams = (hipLaunchParams*) malloc(sizeof(hipLaunchParams) * ngpus);
    hipStream_t     *streams      = (hipStream_t*)     malloc(sizeof(hipStream_t)     * ngpus);
    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipStreamCreate(&streams[i]));
    }
    CHECK(hipSetDevice(0));

    // set up gpu card
    int *d_u[ngpus];

    size_t isize = 5;
    size_t ibyte = isize * sizeof(int);

    int *host_ref = (int *) calloc(isize * ngpus, sizeof(int));
    int *gpu_ref = (int *) calloc(isize * ngpus, sizeof(int));

    for (int i = 0; i < ngpus; i++)
    {
        for (int j = 0; j < isize; j++)
        {
            int idx = i * isize + j;
            host_ref[idx] = idx;
        }
    }

    printf("HOST:\n");
    for (int d = 0; d < ngpus; d++) {
        for (int i = 0; i < isize; i++) {
            printf("%d ", host_ref[d * isize + i]);
        }
        printf("\n");
    }

    dim3 block(isize);
    dim3 grid(1);

    // TODO: parallelize
    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipMalloc((void **) &d_u[i], ibyte));
        CHECK(hipMemcpyAsync(d_u[i], &host_ref[i * isize], ibyte, hipMemcpyHostToDevice, streams[i]));
    }
    CHECK(hipSetDevice(0));

    void *args[ngpus][1];

    for (int i = 0; i < ngpus; i++)
    {
        args[i][0] = &d_u[i];
        launchParams[i].func = (void*)test;
        launchParams[i].gridDim = grid;
        launchParams[i].blockDim = block;
        launchParams[i].sharedMem = 0;
        launchParams[i].stream = streams[i];
        launchParams[i].args = args[i];
    }

    hipLaunchCooperativeKernelMultiDevice(launchParams, ngpus);

    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipMemcpyAsync(&gpu_ref[isize * i], d_u[i], ibyte, hipMemcpyDeviceToHost, streams[i]));
    }

    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipStreamDestroy(streams[i]));
        CHECK(hipFree(d_u[i]));
    }

    printf("DEVICE:\n");
    for (int d = 0; d < ngpus; d++) {
        for (int i = 0; i < isize; i++) {
            printf("%d ", gpu_ref[d * isize + i]);
        }
        printf("\n");
    }

    for (int i = 0; i < ngpus; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipDeviceReset());
    }

    free(launchParams);
    free(streams);

    free(host_ref);
    free(gpu_ref);
}
