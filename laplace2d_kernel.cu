#include "hip/hip_runtime.h"
#include "constants.h"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

__global__ void gpu_laplace2d_base(float* __restrict__ d_u1,
			           float* __restrict__ d_u2,
                                   int jstart,
                                   int jend)
{
    int   i, j,
          tx, ty,
          idx, ioff, joff;
    float u2 = 0.0f, fourth=1.0f/4.0f;

    
    tx = threadIdx.x;
    ty = threadIdx.y;
    i  = tx + blockIdx.x*BLOCK_X;
    j  = ty + blockIdx.y*BLOCK_Y;// + jstart;
    ioff = 1;
    joff = NX;

    idx = i + j *joff;
    if (i>=0 && i<=NX-1 && j>=jstart && j<=jend) {
        if (i==0 || i==NX-1 || j==jstart || j==jend)
          u2 = d_u1[idx]; // Dirichlet b.c.'s
        else {
          u2 = (d_u1[idx-ioff]  +
                d_u1[idx+ioff]  +
                d_u1[idx-joff]  +
                d_u1[idx+joff]) * fourth;
        }
        d_u2[idx] = u2;
    }
}

__global__ void gpu_laplace2d_smem(float* __restrict__ d_u1,
			           float* __restrict__ d_u2,
                                   int jstart,
                                   int jend)
{
    int   i, j,
          tx, ty, sx, sy,
          idx, ioff, joff;
    float u2 = 0.0f, fourth=1.0f/4.0f;

    tx = threadIdx.x;
    ty = threadIdx.y;
    sx = tx+1;
    sy = ty+1;
    i  = tx + blockIdx.x*BLOCK_X;
    j  = ty + blockIdx.y*BLOCK_Y;
    ioff = 1;
    joff = NX;

    grid_group g = this_grid();
    thread_block tb = this_thread_block();
    __shared__ float smem[BLOCK_Y+2][BLOCK_X+2];

    idx = i + j*joff;
    if (i != 0)           smem[sy][sx-1]   = d_u1[idx-ioff];
    if (i != NX-1)        smem[sy][sx+1]   = d_u1[idx+ioff];
    if (j != 0)           smem[sy-1][sx]   = d_u1[idx-joff];
    if (j != NY-1)        smem[sy+1][sx]   = d_u1[idx+joff];
    smem[sy][sx] = d_u1[idx];
    tb.sync();
    if (i>=0 && i<=NX-1 && j>=jstart && j<=jend) {
        if (i==0 || i==NX-1 || j==jstart || j==jend)
          u2 = d_u1[idx]; // Dirichlet b.c.'s
        else {
          u2 = (smem[sy][sx-1]  +
                smem[sy][sx+1]  +
                smem[sy-1][sx]  +
                smem[sy+1][sx]) * fourth;
        }
        d_u2[idx] = u2;
    }
}

__global__ void gpu_laplace2d_coop(float* __restrict__ d_u1,
			      float* __restrict__ d_u2)
{
    int   i, j, q, x, y,
          bx, by,
          gx, gy,
          xskip, yskip, 
          idx, ioff, joff;
    float u2, *d_tmp, fourth=1.0f/4.0f;
    
    bx = blockDim.x;
    by = blockDim.y;
    i  = threadIdx.x + blockIdx.x*bx;
    j  = threadIdx.y + blockIdx.y*by;
    gx = gridDim.x;
    gy = gridDim.y;

    xskip = bx * gx;
    yskip = by * gy;
    
    ioff = 1;
    joff = NX;
    grid_group g = this_grid();
    
    for (q = 1; q <= ITERATIONS; q++) {
        for (y=j; y<NY; y+=yskip) {
            for (x=i; x<NX; x+=xskip) {
                idx = x + y*joff;
                if (x==0 || x==NX-1 || y==0 || y==NY-1)
                  u2 = d_u1[idx]; // Dirichlet b.c.'s
                else {
                  u2 = (d_u1[idx-ioff]  +
                        d_u1[idx+ioff]  +
                        d_u1[idx-joff]  +
                        d_u1[idx+joff]) * fourth;    
                }
                d_u2[idx] = u2;
            }
        }
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
        g.sync();
    }
}

__global__ void gpu_laplace2d_coop_multi_gpu(float* d_u1,
			                     float* d_u2,
                                             int dev,
			                     float* d_u3
                                             )
{
    int   i, j, q, x, y,
          xskip, yskip, 
          idx;
    float u2, *d_tmp, fourth=1.0f/4.0f;
    
    i  = threadIdx.x + blockIdx.x*BLOCK_X;
    j  = threadIdx.y + blockIdx.y*BLOCK_Y;

    xskip = BLOCK_X * gridDim.x;
    yskip = BLOCK_Y * gridDim.y;

    multi_grid_group mg = this_multi_grid();

    // TODO: Inter-grid syncronization with async memcpy!
    
    for (q = 1; q <= ITERATIONS; q++) {
        for (y=j; y<NY; y+=yskip) {
            for (x=i; x<NX; x+=xskip) {
                if (x>=0 && x<=NX-1 && y>=1 && y<=NY/NGPUS) {
                    idx = x + y*NX;
                    if (x==0 || x==NX-1 || y==1 || y==NY/NGPUS)
                      u2 = d_u1[idx]; // Dirichlet b.c.'s
                    else {
                      u2 = (d_u1[idx-1]  +
                            d_u1[idx+1]  +
                            d_u1[idx-NX]  +
                            d_u1[idx+NX]) * fourth;    
                    }
                    d_u2[idx] = u2;
                }
            }
        }
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
        mg.sync();
    }
}

__global__ void gpu_laplace2d_coop_smem(float* __restrict__ d_u1,
			      float* __restrict__ d_u2)
{
    int   i, j, q, x, y,
          tx, ty, sx, sy,
          xskip, yskip, 
          idx, ioff, joff;
    float u2, *d_tmp, fourth=1.0f/4.0f;

    tx = threadIdx.x;
    ty = threadIdx.y;
    sx = tx+1;
    sy = ty+1;
    i  = tx + blockIdx.x*BLOCK_X;
    j  = ty + blockIdx.y*BLOCK_Y;
    xskip = BLOCK_X * gridDim.x;
    yskip = BLOCK_Y * gridDim.y;
    
    ioff = 1;
    joff = NX;
    grid_group g = this_grid();
    thread_block tb = this_thread_block();

    __shared__ float smem[BLOCK_Y+2][BLOCK_X+2];

    for (q = 1; q <= ITERATIONS; q++) {
        for (y=j; y<NY; y+=yskip) {
            for (x=i; x<NX; x+=xskip) {
                idx = x + y*joff;
                tb.sync();
                if (x != 0)           smem[sy][sx-1]   = d_u1[idx-ioff];
                if (x != NX-1)        smem[sy][sx+1]   = d_u1[idx+ioff];
                if (y != 0)           smem[sy-1][sx]   = d_u1[idx-joff];
                if (y != NY-1)        smem[sy+1][sx]   = d_u1[idx+joff];
                smem[sy][sx] = d_u1[idx];
                tb.sync();

                if (x==0 || x==NX-1 || y==0 || y==NY-1) {
                  u2 = d_u1[idx]; // Dirichlet b.c.'s
                }
                else {
                  u2 = (smem[sy][sx-1]  +
                        smem[sy][sx+1]  +
                        smem[sy-1][sx]  +
                        smem[sy+1][sx]) * fourth;
                }
                d_u2[idx] = u2;
            }
        }
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
        g.sync();
    }
}
