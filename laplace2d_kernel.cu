#include "hip/hip_runtime.h"
#include "constants.h"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

__global__ void gpu_laplace2d_base(float* __restrict__ d_u1,
			           float* __restrict__ d_u2,
                                   int jstart,
                                   int jend)
{
    int   i, j,
          tx, ty,
          idx, ioff, joff;
    float u2 = 0.0f, fourth=1.0f/4.0f;

    
    tx = threadIdx.x;
    ty = threadIdx.y;
    i  = tx + blockIdx.x*BLOCK_X;
    j  = ty + blockIdx.y*BLOCK_Y;// + jstart;
    ioff = 1;
    joff = NX;

    idx = i + j *joff;
    if (i>=0 && i<=NX-1 && j>=jstart && j<=jend) {
        if (i==0 || i==NX-1 || j==jstart || j==jend)
          u2 = d_u1[idx]; // Dirichlet b.c.'s
        else {
          u2 = (d_u1[idx-ioff]  +
                d_u1[idx+ioff]  +
                d_u1[idx-joff]  +
                d_u1[idx+joff]) * fourth;
        }
        d_u2[idx] = u2;
    }
}

__global__ void gpu_laplace2d_smem(float* __restrict__ d_u1,
			           float* __restrict__ d_u2,
                                   int jstart,
                                   int jend)
{
    int   i, j,
          tx, ty, sx, sy,
          idx, ioff, joff;
    float u2 = 0.0f, fourth=1.0f/4.0f;

    tx = threadIdx.x;
    ty = threadIdx.y;
    sx = tx+1;
    sy = ty+1;
    i  = tx + blockIdx.x*BLOCK_X;
    j  = ty + blockIdx.y*BLOCK_Y;
    ioff = 1;
    joff = NX;

    grid_group g = this_grid();
    thread_block tb = this_thread_block();
    __shared__ float smem[BLOCK_Y+2][BLOCK_X+2];

    idx = i + j*joff;
    if (i != 0)           smem[sy][sx-1]   = d_u1[idx-ioff];
    if (i != NX-1)        smem[sy][sx+1]   = d_u1[idx+ioff];
    if (j != 0)           smem[sy-1][sx]   = d_u1[idx-joff];
    if (j != NY-1)        smem[sy+1][sx]   = d_u1[idx+joff];
    smem[sy][sx] = d_u1[idx];
    tb.sync();
    if (i>=0 && i<=NX-1 && j>=jstart && j<=jend) {
        if (i==0 || i==NX-1 || j==jstart || j==jend)
          u2 = d_u1[idx]; // Dirichlet b.c.'s
        else {
          u2 = (smem[sy][sx-1]  +
                smem[sy][sx+1]  +
                smem[sy-1][sx]  +
                smem[sy+1][sx]) * fourth;
        }
        d_u2[idx] = u2;
    }
}

__global__ void gpu_laplace2d_coop(float* __restrict__ d_u1,
			      float* __restrict__ d_u2)
{
    int   i, j, q, x, y,
          bx, by,
          gx, gy,
          xskip, yskip, 
          idx, ioff, joff;
    float u2, *d_tmp, fourth=1.0f/4.0f;
    
    bx = blockDim.x;
    by = blockDim.y;
    i  = threadIdx.x + blockIdx.x*bx;
    j  = threadIdx.y + blockIdx.y*by;
    gx = gridDim.x;
    gy = gridDim.y;

    xskip = bx * gx;
    yskip = by * gy;
    
    ioff = 1;
    joff = NX;
    grid_group g = this_grid();
    
    for (q = 1; q <= ITERATIONS; q++) {
        for (y=j; y<NY; y+=yskip) {
            for (x=i; x<NX; x+=xskip) {
                idx = x + y*joff;
                if (x==0 || x==NX-1 || y==0 || y==NY-1)
                  u2 = d_u1[idx]; // Dirichlet b.c.'s
                else {
                  u2 = (d_u1[idx-ioff]  +
                        d_u1[idx+ioff]  +
                        d_u1[idx-joff]  +
                        d_u1[idx+joff]) * fourth;    
                }
                d_u2[idx] = u2;
            }
        }
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
        g.sync();
    }
}

__global__ void gpu_laplace2d_coop_multi_gpu(float* d_u1,
			                     float* d_u2,
			                     float* d_u3,
                                             int dev,
                                             int jstart,
                                             int jend
                                             )
{
    int   i, j, q, x, y,
          idx;
    float u2, *d_tmp, fourth=1.0f/4.0f;
    
    x  = threadIdx.x + blockIdx.x*BLOCK_X;
    y  = threadIdx.y + blockIdx.y*BLOCK_Y;

    multi_grid_group mg = this_multi_grid();
    
    for (q = 1; q <= ITERATIONS; q++) {
        if (dev==0) memcpy(d_u1 + (NY/NGPUS + 1) * NX, d_u3 + NX, NX*sizeof(float));
        else if (dev==NGPUS-1) memcpy(d_u1, d_u3 + (NY/NGPUS) * NX, NX*sizeof(float));
        else {
            memcpy(d_u1 + (NY/NGPUS + 1) * NX, d_u3 + NX, NX*sizeof(float));
            memcpy(d_u1, d_u3 + (NY/NGPUS) * NX, NX*sizeof(float));
        }
        //for (y=j; y<NY; y+=BLOCK_Y * gridDim.y) {
        //    for (x=i; x<NX; x+=BLOCK_X * gridDim.x) {
                if (x>=0 && x<=NX-1 && y>=jstart && y<=jend) {
                    idx = x + y*NX;
                    if (x==0 || x==NX-1 || y==jstart || y==jend)
                      u2 = d_u1[idx]; // Dirichlet b.c.'s
                    else {
                      u2 = (d_u1[idx-1]  +
                            d_u1[idx+1]  +
                            d_u1[idx-NX]  +
                            d_u1[idx+NX]) * fourth;    
                    }
                    d_u2[idx] = u2;
                }
        //    }
        //}
        //mg.sync();
        mg.sync();
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
    }
}

__global__ void gpu_laplace2d_coop_smem(float* __restrict__ d_u1,
			      float* __restrict__ d_u2)
{
    int   i, j, q, x, y,
          tx, ty, sx, sy,
          xskip, yskip, 
          idx, ioff, joff;
    float u2, *d_tmp, fourth=1.0f/4.0f;

    tx = threadIdx.x;
    ty = threadIdx.y;
    sx = tx+1;
    sy = ty+1;
    i  = tx + blockIdx.x*BLOCK_X;
    j  = ty + blockIdx.y*BLOCK_Y;
    xskip = BLOCK_X * gridDim.x;
    yskip = BLOCK_Y * gridDim.y;
    
    ioff = 1;
    joff = NX;
    grid_group g = this_grid();
    thread_block tb = this_thread_block();

    __shared__ float smem[BLOCK_Y+2][BLOCK_X+2];

    for (q = 1; q <= ITERATIONS; q++) {
        for (y=j; y<NY; y+=yskip) {
            for (x=i; x<NX; x+=xskip) {
                idx = x + y*joff;
                tb.sync();
                if (x != 0)           smem[sy][sx-1]   = d_u1[idx-ioff];
                if (x != NX-1)        smem[sy][sx+1]   = d_u1[idx+ioff];
                if (y != 0)           smem[sy-1][sx]   = d_u1[idx-joff];
                if (y != NY-1)        smem[sy+1][sx]   = d_u1[idx+joff];
                smem[sy][sx] = d_u1[idx];
                tb.sync();

                if (x==0 || x==NX-1 || y==0 || y==NY-1) {
                  u2 = d_u1[idx]; // Dirichlet b.c.'s
                }
                else {
                  u2 = (smem[sy][sx-1]  +
                        smem[sy][sx+1]  +
                        smem[sy-1][sx]  +
                        smem[sy+1][sx]) * fourth;
                }
                d_u2[idx] = u2;
            }
        }
        d_tmp = d_u1; d_u1 = d_u2; d_u2 = d_tmp; // swap d_u1 and d_u2
        g.sync();
    }
}
