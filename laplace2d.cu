#include <stdio.h>
#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_initializer.h"
#include "laplace2d_error_checker.h"
#include "laplace2d_utils.h"
#include "laplace2d_timer.cu"
#include "laplace2d_dispatch.cu"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

int main(int argc, const char **argv){
    int    ibyte = NX*NY * sizeof(float);
    float  *h_u1, *h_u2,
           *d_u1, *d_u2,
           milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_u1 = (float *)malloc(ibyte);
    h_u2 = (float *)malloc(ibyte);
    CU(hipMalloc((void **)&d_u1, ibyte));
    CU(hipMalloc((void **)&d_u2, ibyte));

    hipStream_t streams[STREAMS];

    for (int i = 0; i < STREAMS; i++)
    {
        CU(hipStreamCreate( &streams[i] ));
    }

    print_program_info();

    initialize_host_region(h_u1);

    start_timer(start);
    CU(hipMemcpy(d_u1, h_u1, ibyte, hipMemcpyHostToDevice));
    stop_timer(&start, &stop, &milli, "\ncudaMemcpyHostToDevice: %.1f (ms) \n");

    readSolution(h_u1);

    start_timer(start);
    //if (COOP) dispatch_cooperative_groups_kernels(d_u1, d_u2);
    //else
    dispatch_kernels(d_u1, d_u2, streams);
    stop_timer(&start, &stop, &milli, "\nKernel execution time: %.1f (ms) \n");
    
    // TODO: Trenger async memcpy, ettersom det er bare stream 0 som copies tilbake
    // Dette burde vel ikke være noe problem, ettersom det er på samme device...
    start_timer(start);
    CU(hipMemcpy(h_u2, d_u1, ibyte, hipMemcpyDeviceToHost));
    stop_timer(&start, &stop, &milli, "\ncudaMemcpyDeviceToHost: %.1f (ms) \n");

    check_domain_errors(h_u1, h_u2, NX, NY);

    if (DEBUG) print_corners(h_u1, h_u2);
    if (TEST || DEBUG) saveResult(h_u2);

    for (int i = 0; i < STREAMS; i++)
    {
        CU(hipStreamDestroy(streams[i]));
    }

    CU(hipFree(d_u1));
    CU(hipFree(d_u2));
    free(h_u1);
    free(h_u2);

    hipDeviceReset();
}
