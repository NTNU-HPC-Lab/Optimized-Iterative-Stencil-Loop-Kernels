#include <stdio.h>
#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_initializer.h"
#include "laplace2d_error_checker.h"
#include "laplace2d_utils.h"
#include "laplace2d_timer.cu"
#include "laplace2d_dispatch.cu"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

int main(int argc, const char **argv){
    float  *h_u1, *h_u2,
           *d_u1[NGPUS], *d_u2[NGPUS],
           milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_u1 = (float *)malloc(BYTES);
    h_u2 = (float *)malloc(BYTES);

    for (int i = 0; i < NGPUS; i++)
    {
        CU(hipMalloc((void **)&d_u1[i], BYTES_PER_GPU));
        CU(hipMalloc((void **)&d_u2[i], BYTES_PER_GPU));
    }

    hipStream_t streams[NGPUS];

    for (int i = 0; i < NGPUS; i++)
    {
        CU(hipStreamCreate( &streams[i] ));
    }

    print_program_info();

    initialize_host_region(h_u1);

    for (int i = 0; i < NGPUS; i++)
    {
        CU(hipMemcpyAsync(d_u1[i], &h_u1[i * OFFSET], BYTES_PER_GPU, hipMemcpyHostToDevice, streams[i]));
    }

    readSolution(h_u1);

    start_timer(start);
    //if (COOP) dispatch_cooperative_groups_kernels(d_u1, d_u2);
    //else
    dispatch_kernels(d_u1, d_u2, streams);
    stop_timer(&start, &stop, &milli, "\nKernel execution time: %.1f (ms) \n");
    
    for (int i = 0; i < NGPUS; i++)
    {
        CU(hipMemcpyAsync(&h_u2[i * OFFSET], d_u1[i], BYTES_PER_GPU, hipMemcpyDeviceToHost, streams[i]));
    }
    
    for (int i = 0; i < NGPUS; i++)
    {
        hipDeviceSynchronize();
    }

    check_domain_errors(h_u1, h_u2, NX, NY);

    if (DEBUG) print_corners(h_u1, h_u2);
    if (TEST || DEBUG) saveResult(h_u2);

    for (int i = 0; i < NGPUS; i++)
    {
        CU(hipStreamDestroy(streams[i]));
        CU(hipFree(d_u1[i]));
        CU(hipFree(d_u2[i]));
    }

    free(h_u1);
    free(h_u2);

    hipDeviceReset();
}
