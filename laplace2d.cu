#include <stdio.h>
#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_initializer.h"
#include "laplace2d_error_checker.h"
#include "laplace2d_utils.h"
#include "laplace2d_timer.cu"
#include "laplace2d_dispatch.cu"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

/*
 * enable P2P memcopies between GPUs (all GPUs must be compute capability 2.0 or
 * later (Fermi or later))
 */
inline void enableP2P (int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        CU(hipSetDevice(i));

        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;

            int peer_access_available = 0;
            CU(hipDeviceCanAccessPeer(&peer_access_available, i, j));

            if (peer_access_available) CU(hipDeviceEnablePeerAccess(j, 0));
        }
    }
}

int main(int argc, const char **argv){
    float  *h_ref, *d_ref,
           *d_u1[NGPUS], *d_u2[NGPUS];//,
           //milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_ref = (float *)malloc(BYTES);
    if (hipHostMalloc((void**)&d_ref, BYTES) != hipSuccess) {
        fprintf(stderr, "Error returned from pinned host memory allocation\n");
        exit(1);
    }

    //enableP2P(NGPUS);

    print_program_info();

    initialize_host_region(d_ref);

    for (int i = 0; i < NGPUS; i++)
    {
        hipSetDevice(i);
        CU(hipMalloc((void **)&d_u1[i], BYTES_PER_GPU + BYTES_BORDER));
        CU(hipMalloc((void **)&d_u2[i], BYTES_PER_GPU + BYTES_BORDER));
    }

    hipStream_t streams[NGPUS];

    for (int i = 0; i < NGPUS; i++)
    {
        hipSetDevice(i);
        CU(hipStreamCreate( &streams[i] ));
    }

    for (int i = 0; i < NGPUS; i++)
    {
        hipSetDevice(i);
        CU(hipMemcpyAsync(&d_u1[i][NX], &d_ref[i * OFFSET], BYTES_PER_GPU, hipMemcpyHostToDevice, streams[i]));
    }

    readSolution(h_ref);

    //start_timer(start);
    //stop_timer(&start, &stop, &milli, "\nKernel execution time: %.1f (ms) \n");
    //if (COOP) dispatch_cooperative_groups_kernels(d_u1, d_u2);
    //else
    dispatch_kernels(d_u1, d_u2, streams);
    
    for (int i = 0; i < NGPUS; i++)
    {
        hipSetDevice(i);
        CU(hipMemcpyAsync(&d_ref[i * OFFSET], &d_u1[i][NX], BYTES_PER_GPU, hipMemcpyDeviceToHost, streams[i]));
    }
    
    for (int i = 0; i < NGPUS; i++)
    {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }

    check_domain_errors(h_ref, d_ref, NX, NY);

    if (DEBUG) print_corners(h_ref, d_ref);
    if (TEST || DEBUG) saveResult(d_ref);

    for (int i = 0; i < NGPUS; i++)
    {
        hipSetDevice(i);
        CU(hipStreamDestroy(streams[i]));
        CU(hipFree(d_u1[i]));
        CU(hipFree(d_u2[i]));
    }

    hipHostFree(d_ref);
    free(h_ref);

    hipDeviceReset();
}
