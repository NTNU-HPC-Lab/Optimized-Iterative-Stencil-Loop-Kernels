#include <stdio.h>
#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_initializer.h"
#include "laplace2d_error_checker.h"
#include "laplace2d_utils.h"
#include "laplace2d_dispatch.cu"
#include "omp.h"

int main(int argc, const char **argv) {
    float  *h_ref, *d_ref,
           *d_u1[NGPUS], *d_u2[NGPUS],
           milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    print_program_info();

    if (DEBUG) {
        h_ref = (float *)malloc(BYTES);
        readSolution(h_ref);
    }

    hipEventRecord(start);

    hipStream_t streams[NGPUS];
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipStreamCreate( &streams[i] ));
    }

    if (NGPUS>1) ENABLE_P2P(NGPUS);

    if (hipHostMalloc((void**)&d_ref, BYTES) != hipSuccess) {
        fprintf(stderr, "Error returned from pinned host memory allocation\n");
        exit(1);
    }

    initialize_host_region(d_ref);

    unsigned long size = BYTES_PER_GPU;
    if (NGPUS>1) size += HALO_BYTES;
#pragma omp parallel for num_threads(NGPUS)
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMalloc((void **)&d_u1[i], size));
        CU(hipMalloc((void **)&d_u2[i], size));
    }

    int offset;
    if (NGPUS==1) offset=0;
    else          offset=HALO_DEPTH * NX;
#pragma omp parallel for num_threads(NGPUS)
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMemcpyAsync(&d_u1[i][offset], &d_ref[i * OFFSET], BYTES_PER_GPU, hipMemcpyHostToDevice, streams[i]));
    }

    if(NGPUS==1) {
        if (COOP) dispatch_cooperative_groups_kernels(d_u1[0], d_u2[0]);
        else      dispatch_kernels(d_u1[0], d_u2[0]);
    } else dispatch_multi_gpu_kernels(d_u1, d_u2, streams);
    
#pragma omp parallel for num_threads(NGPUS)
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMemcpyAsync(&d_ref[i * OFFSET], &d_u1[i][offset], BYTES_PER_GPU, hipMemcpyDeviceToHost, streams[i]));
    }
    
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }
    
    hipSetDevice(0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    if (DEBUG) {
        //print_corners(h_ref, d_ref);
        check_domain_errors(h_ref, d_ref, NX, NY);
        //saveResult(d_ref);
        free(h_ref);
    }
    printf("%.4f\n", milli); // Print time spent in ms

    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipStreamDestroy(streams[i]));
        CU(hipFree(d_u1[i]));
        CU(hipFree(d_u2[i]));
        hipDeviceReset();
    }
    hipHostFree(d_ref);
}
