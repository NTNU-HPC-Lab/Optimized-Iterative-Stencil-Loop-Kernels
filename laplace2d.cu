#include <stdio.h>
#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_initializer.h"
#include "laplace2d_error_checker.h"
#include "laplace2d_utils.h"
#include "laplace2d_timer.cu"
#include "laplace2d_dispatch.cu"
#include "hip/hip_cooperative_groups.h"
#include "omp.h"
using namespace cooperative_groups;

int main(int argc, const char **argv) {
    float  *h_ref, *d_ref,
           *d_u1[NGPUS], *d_u2[NGPUS],
           milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_ref = (float *)malloc(BYTES);
    if (hipHostMalloc((void**)&d_ref, BYTES) != hipSuccess) {
        fprintf(stderr, "Error returned from pinned host memory allocation\n");
        exit(1);
    }

    if (NGPUS>1) ENABLE_P2P(NGPUS);

    if (DEBUG) {
        print_program_info();
        initialize_host_region(d_ref);
    }

    int size = BYTES_PER_GPU;
    if (NGPUS>1) size+=BYTES_HALO;
#pragma omp parallel for
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMalloc((void **)&d_u1[i], size));
        CU(hipMalloc((void **)&d_u2[i], size));
    }

    hipSetDevice(0);
    start_timer(start);

    int offset;
    if (NGPUS==1) offset=0;
    else          offset=NX;
#pragma omp parallel for
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMemcpy(&d_u1[i][offset], &d_ref[i * OFFSET], BYTES_PER_GPU, hipMemcpyHostToDevice));
    }

    if (DEBUG) {
        readSolution(h_ref);
    }

    if(NGPUS==1) {
        if (COOP) dispatch_cooperative_groups_kernels(d_u1[0], d_u2[0]);
        else      dispatch_kernels(d_u1[0], d_u2[0]);
    } else dispatch_multi_gpu_kernels(d_u1, d_u2);
    
#pragma omp parallel for
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMemcpy(&d_ref[i * OFFSET], &d_u1[i][offset], BYTES_PER_GPU, hipMemcpyDeviceToHost));
    }
    
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }

    hipSetDevice(0);
    stop_timer(&start, &stop, &milli, "%.4f\n");

    if (DEBUG) {
        print_corners(h_ref, d_ref);
        check_domain_errors(h_ref, d_ref, NX, NY);
        saveResult(d_ref);
    }

    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipFree(d_u1[i]));
        CU(hipFree(d_u2[i]));
    }

    hipHostFree(d_ref);
    free(h_ref);

    hipDeviceReset();
}
