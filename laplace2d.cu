#include <stdio.h>
#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_initializer.h"
#include "laplace2d_error_checker.h"
#include "laplace2d_utils.h"
#include "laplace2d_timer.cu"
#include "laplace2d_dispatch.cu"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

int main(int argc, const char **argv) {
    float  *h_ref, *d_ref,
           *d_u1[NGPUS], *d_u2[NGPUS],
           milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_ref = (float *)malloc(BYTES);
    if (hipHostMalloc((void**)&d_ref, BYTES) != hipSuccess) {
        fprintf(stderr, "Error returned from pinned host memory allocation\n");
        exit(1);
    }

    if (NGPUS>1) ENABLE_P2P(NGPUS);

    if (DEBUG) {
        print_program_info();
        initialize_host_region(d_ref);
    }

    int size = BYTES_PER_GPU;
    if (NGPUS>1) size+=BYTES_HALO;
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMalloc((void **)&d_u1[i], size));
        CU(hipMalloc((void **)&d_u2[i], size));
    }

    hipStream_t streams[NGPUS];

    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipStreamCreate( &streams[i] ));
    }

    hipSetDevice(0);
    start_timer(start);

    int offset;
    if (NGPUS==1) offset=0;
    else          offset=NX;
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMemcpyAsync(&d_u1[i][offset], &d_ref[i * OFFSET], BYTES_PER_GPU, hipMemcpyHostToDevice, streams[i]));
    }

    if (DEBUG) {
        readSolution(h_ref);
    }

    if(NGPUS==1) {
        if (COOP) dispatch_cooperative_groups_kernels(d_u1[0], d_u2[0]);
        else      dispatch_kernels(d_u1[0], d_u2[0]);
    } else dispatch_multi_gpu_kernels(d_u1, d_u2, streams);
    
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipMemcpyAsync(&d_ref[i * OFFSET], &d_u1[i][offset], BYTES_PER_GPU, hipMemcpyDeviceToHost, streams[i]));
    }
    
    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }

    hipSetDevice(0);
    //stop_timer(&start, &stop, &milli, "\nTotal execution time: %.1f (ms) \n");
    stop_timer(&start, &stop, &milli, "%.1f\n");

    if (DEBUG) {
        print_corners(h_ref, d_ref);
        check_domain_errors(h_ref, d_ref, NX, NY);
        saveResult(d_ref);
    }

    for (int i = 0; i < NGPUS; i++) {
        hipSetDevice(i);
        CU(hipStreamDestroy(streams[i]));
        CU(hipFree(d_u1[i]));
        CU(hipFree(d_u2[i]));
    }

    hipHostFree(d_ref);
    free(h_ref);

    hipDeviceReset();
}
