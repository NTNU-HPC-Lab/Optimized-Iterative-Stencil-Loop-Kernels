#include <stdio.h>
#include "hip/hip_runtime_api.h"
#include "laplace2d_timer.h"
#include "laplace2d_utils.cu"
#include "laplace2d_kernel.cu"
#include "laplace2d_initializer.h"
#include "laplace2d_error_checker.h"
//#include "hip/hip_cooperative_groups.h"
//using namespace cooperative_groups;

#define CU checkCudaErrors 
#define start_timer hipEventRecord

#ifndef BLOCK_X
#define BLOCK_X 128
#endif

#ifndef BLOCK_Y
#define BLOCK_Y 4
#endif

#define NX 256
#define NY 256

#define ITERATIONS 8096

void cpu_laplace2d(int nx, int ny, float* h_u1, float* h_u2);

int main(int argc, const char **argv){
    int    i, j, ind,
           ibyte = NX*NY * sizeof(float);
    float  *h_u1, *h_u2, *h_u3, *h_swap,
           *d_u1, *d_u2, *d_tmp,
           milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_u1 = (float *)malloc(ibyte);
    h_u2 = (float *)malloc(ibyte);
    h_u3 = (float *)malloc(ibyte);
    CU(hipMalloc((void **)&d_u1, ibyte));
    CU(hipMalloc((void **)&d_u2, ibyte));

    initialize_host_region(h_u1);

    start_timer(start);
    CU(hipMemcpy(d_u1, h_u1, ibyte, hipMemcpyHostToDevice));
    stop_timer(&start, &stop, &milli, "\ncudaMemcpyHostToDevice: %.1f (ms) \n");

    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(
        1 + (NX-1)/BLOCK_X,
        1 + (NY-1)/BLOCK_Y
    );

    int nx = NX;
    int ny = NY;
    int iter = ITERATIONS;
    
    void *args[] = {
        &d_u1,
        &d_u2,
        (void *)&nx,
        (void *)&ny,
        (void *)&iter
    };

    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    // initialize, then launch
    printf("Multiprocessorcount: %d, Grid: x: %d y: %d\n", deviceProp.multiProcessorCount, dimGrid.x, dimGrid.y);

    start_timer(start);
    hipLaunchCooperativeKernel((void*)gpu_laplace2d, deviceProp.multiProcessorCount, dimBlock, args);
    stop_timer(&start, &stop, &milli, "\ngpu_laplace2d (cooperative groups): %.1f (ms) \n");
    
    start_timer(start);
    CU(hipMemcpy(h_u2, d_u1, ibyte, hipMemcpyDeviceToHost));
    stop_timer(&start, &stop, &milli, "\ncudaMemcpyDeviceToHost: %.1f (ms) \n");


    start_timer(start);
    for (i = 1; i <= ITERATIONS; ++i) {
        cpu_laplace2d(NX, NY, h_u1, h_u3);
        h_swap = h_u1; h_u1 = h_u3; h_u3 = h_swap;   // swap h_u1 and h_u3
    }
    stop_timer(&start, &stop, &milli, "\ncpu_laplace2d: %.1f (ms) \n");

    check_domain_errors(h_u1, h_u2);

    // print out corner of array
    for (j=0; j<8; j++) {
      for (i=0; i<8; i++) {
        ind = i + j*NX;
        printf(" %5.2f ", h_u2[ind]);
      }
      printf("\n");
    }

    saveResult(h_u1);

    CU(hipFree(d_u1));
    CU(hipFree(d_u2));
    free(h_u1);
    free(h_u2);
    free(h_u3);

    hipDeviceReset();
}
