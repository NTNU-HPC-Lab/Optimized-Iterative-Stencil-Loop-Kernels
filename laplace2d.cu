#include <stdio.h>
#include "constants.h"
#include "hip/hip_runtime_api.h"
#include "laplace2d_initializer.h"
#include "laplace2d_error_checker.h"
#include "laplace2d_utils.h"
#include "laplace2d_timer.cu"
#include "laplace2d_dispatch.cu"
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;

#define CU checkCudaErrors 
#define start_timer hipEventRecord

void print_corners(float *h_u1, float *h_u2) {
    int i, j, ind;
    printf("DEVICE\n");
    for (j=0; j<8; j++) {
      for (i=0; i<8; i++) {
        ind = i + j*NX;
        printf(" %5.2f ", h_u2[ind]);
      }
      printf("\n");
    }

    printf("\n");

    printf("HOST\n");
    for (j=0; j<8; j++) {
      for (i=0; i<8; i++) {
        ind = i + j*NX;
        printf(" %5.2f ", h_u1[ind]);
      }
      printf("\n");
    }
}

int main(int argc, const char **argv){
    int    ibyte = NX*NY * sizeof(float);
    float  *h_u1, *h_u2,
           *d_u1, *d_u2,
           milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_u1 = (float *)malloc(ibyte);
    h_u2 = (float *)malloc(ibyte);
    CU(hipMalloc((void **)&d_u1, ibyte));
    CU(hipMalloc((void **)&d_u2, ibyte));

    initialize_host_region(h_u1);

    start_timer(start);
    CU(hipMemcpy(d_u1, h_u1, ibyte, hipMemcpyHostToDevice));
    stop_timer(&start, &stop, &milli, "\ncudaMemcpyHostToDevice: %.1f (ms) \n");

    readSolution(h_u1);

    start_timer(start);
    dispatch_kernels(d_u1, d_u2);
    stop_timer(&start, &stop, &milli, "\ngpu_laplace2d (base): %.1f (ms) \n");
    
    start_timer(start);
    CU(hipMemcpy(h_u2, d_u1, ibyte, hipMemcpyDeviceToHost));
    stop_timer(&start, &stop, &milli, "\ncudaMemcpyDeviceToHost: %.1f (ms) \n");

    check_domain_errors(h_u1, h_u2, NX, NY);

    if (DEBUG) print_corners(h_u1, h_u2);

    if (TEST) saveResult(h_u2);


    CU(hipFree(d_u1));
    CU(hipFree(d_u2));
    free(h_u1);
    free(h_u2);

    hipDeviceReset();
}
