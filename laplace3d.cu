#include <stdlib.h>
#include <stdio.h>
#include <stdarg.h>
#include <string.h>
#include <math.h>

#define CU checkCudaErrors 
#define start_timer hipEventRecord

#ifndef BLOCK_X
#define BLOCK_X 128
#endif

#ifndef BLOCK_Y
#define BLOCK_Y 128
#endif

#define NX 256
#define NY 256
#define ITERATIONS 10

#include "hip/hip_runtime_api.h"
#include "laplace3d_timer.h"
#include "laplace3d_kernel.cu"
#include "laplace3d_initializer.h"
#include "laplace3d_error_checker.h"

void saveResult(float *d_u)
{
    char fname[15];
    sprintf(fname, "result");

    FILE *fp_snap = fopen(fname, "w");

    fwrite(d_u, sizeof(float), NX * NY, fp_snap);
    printf("Saving %s: nx = %d ny = %d\n", fname, NX, NY);
    fflush(stdout);
    fclose(fp_snap);

    return;
}


void Gold_laplace3d(int nx, int ny, float* h_u1, float* h_u2);

int main(int argc, const char **argv){
    int    i, j, ind,
           ibyte = NX*NY * sizeof(float);
    float  *h_u1, *h_u2, *h_u3, *h_swap,
           *d_u1, *d_u2, *d_foo,
           milli;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_u1 = (float *)malloc(ibyte);
    h_u2 = (float *)malloc(ibyte);
    h_u3 = (float *)malloc(ibyte);
    CU(hipMalloc((void **)&d_u1, ibyte));
    CU(hipMalloc((void **)&d_u2, ibyte));

    initialize_host_region(h_u1);

    start_timer(start);
    CU(hipMemcpy(d_u1, h_u1, ibyte, hipMemcpyHostToDevice));
    stop_timer(&start, &stop, &milli, "\ncudaMemcpyHostToDevice: %.1f (ms) \n");

    dim3 dimBlock(BLOCK_X,BLOCK_Y);
    dim3 dimGrid(
        1 + (NX-1)/BLOCK_X,
        1 + (NY-1)/BLOCK_Y
    );

    start_timer(start);
    for (i = 1; i <= ITERATIONS; ++i) {
      GPU_laplace3d<<<dimGrid, dimBlock>>>(
              d_u1,
              d_u2
      );
      getLastCudaError("GPU_laplace3d execution failed\n");

      d_foo = d_u1; d_u1 = d_u2; d_u2 = d_foo;   // swap d_u1 and d_u2
    }
    stop_timer(&start, &stop, &milli, "\nGPU_laplace3d: %.1f (ms) \n");
    
    start_timer(start);
    CU(hipMemcpy(h_u2, d_u1, ibyte, hipMemcpyDeviceToHost));
    stop_timer(&start, &stop, &milli, "\ncudaMemcpyDeviceToHost: %.1f (ms) \n");


    start_timer(start);
    for (i = 1; i <= ITERATIONS; ++i) {
        Gold_laplace3d(NX, NY, h_u1, h_u3);
        h_swap = h_u1; h_u1 = h_u3; h_u3 = h_swap;   // swap h_u1 and h_u3
    }
    stop_timer(&start, &stop, &milli, "\nCPU_laplace3d: %.1f (ms) \n");

    check_domain_errors(h_u1, h_u2);

    // print out corner of array

    for (j=0; j<8; j++) {
      for (i=0; i<8; i++) {
        ind = i + j*NX;
        printf(" %5.2f ", h_u2[ind]);
      }
      printf("\n");
    }

    saveResult(h_u1);

    CU(hipFree(d_u1));
    CU(hipFree(d_u2));
    free(h_u1);
    free(h_u2);
    free(h_u3);

    hipDeviceReset();
}
